//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: Decision_Predict_terminate.cu
//
// GPU Coder version                    : 2.5
// CUDA/C/C++ source code generated on  : 11-Jun-2024 01:16:03
//

// Include Files
#include "Decision_Predict_terminate.h"
#include "Decision_Predict.h"
#include "Decision_Predict_data.h"
#include "stdio.h"

// Function Definitions
//
// Arguments    : void
// Return Type  : void
//
void Decision_Predict_terminate()
{
  hipError_t errCode;
  errCode = hipGetLastError();
  if (errCode != hipSuccess) {
    fprintf(stderr, "ERR[%d] %s:%s\n", errCode, hipGetErrorName(errCode),
            hipGetErrorString(errCode));
    exit(errCode);
  }
  Decision_Predict_free();
  isInitialized_Decision_Predict = false;
}

//
// File trailer for Decision_Predict_terminate.cu
//
// [EOF]
//
