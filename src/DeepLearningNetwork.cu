#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: DeepLearningNetwork.cu
//
// GPU Coder version                    : 2.5
// CUDA/C/C++ source code generated on  : 11-Jun-2024 01:16:03
//

// Include Files
#include "DeepLearningNetwork.h"
#include "Decision_Predict_internal_types.h"
#include "MWCNNLayer.hpp"
#include "MWCudnnTargetNetworkImpl.hpp"
#include "MWElementwiseAffineLayer.hpp"
#include "MWFCLayer.hpp"
#include "MWFusedConvActivationLayer.hpp"
#include "MWInputLayer.hpp"
#include "MWMaxPoolingLayer.hpp"
#include "MWOutputLayer.hpp"
#include "MWSoftmaxLayer.hpp"
#include "MWTensor.hpp"
#include "MWTensorBase.hpp"
#include "stdio.h"
#include <cstdlib>

// Named Constants
const char *errorString{
    "Abnormal termination due to: %s.\nError in %s (line %d)."};

const char *errStringBase{
    "Error during execution of the generated code. %s at line: %d, file: "
    "%s\nExiting program execution ...\n"};

// Function Declarations
static void checkCleanupCudaError(hipError_t errCode, const char *file,
                                  unsigned int b_line);

static void checkRunTimeError(const char *errMsg, const char *file,
                              unsigned int b_line);

namespace coder {
namespace internal {
static void DeepLearningNetwork_callDelete(decnet0_0 *obj);

}
} // namespace coder

// Function Definitions
//
// Arguments    : void
// Return Type  : void
//
void decnet0_0::allocate()
{
  targetImpl->allocate(275598, 2);
  for (int idx{0}; idx < 10; idx++) {
    layers[idx]->allocate();
  }
  (static_cast<MWTensor<float> *>(inputTensors[0]))
      ->setData(layers[0]->getLayerOutput(0));
}

//
// Arguments    : void
// Return Type  : void
//
void decnet0_0::cleanup()
{
  deallocate();
  for (int idx{0}; idx < 10; idx++) {
    layers[idx]->cleanup();
  }
  if (targetImpl) {
    targetImpl->cleanup();
  }
  isInitialized = false;
  checkCleanupCudaError(hipGetLastError(), __FILE__, __LINE__);
}

//
// Arguments    : void
// Return Type  : void
//
void decnet0_0::deallocate()
{
  targetImpl->deallocate();
  for (int idx{0}; idx < 10; idx++) {
    layers[idx]->deallocate();
  }
}

//
// Arguments    : void
// Return Type  : void
//
void decnet0_0::postsetup()
{
  targetImpl->postSetup(layers, numLayers);
}

//
// Arguments    : void
// Return Type  : void
//
void decnet0_0::resetState()
{
}

//
// Arguments    : void
// Return Type  : void
//
void decnet0_0::setSize()
{
  for (int idx{0}; idx < 10; idx++) {
    layers[idx]->propagateSize();
  }
  allocate();
  postsetup();
}

//
// Arguments    : void
// Return Type  : void
//
void decnet0_0::setup()
{
  if (isInitialized) {
    resetState();
  } else {
    targetImpl->preSetup();
    targetImpl->setAutoTune(true);
    (static_cast<MWInputLayer *>(layers[0]))
        ->createInputLayer(targetImpl, inputTensors[0], "SSCB", 0);
    (static_cast<MWElementwiseAffineLayer *>(layers[1]))
        ->createElementwiseAffineLayer(
            targetImpl, layers[0]->getOutputTensor(0), 1, 1, 18, 1, 1, 18,
            false, 1, 1,
            "./codegen/lib/Decision_Predict/cnn_decnet0_0_imageinput_scale.bin",
            "./codegen/lib/Decision_Predict/"
            "cnn_decnet0_0_imageinput_offset.bin",
            "SSCB", 0);
    (static_cast<MWFusedConvActivationLayer *>(layers[2]))
        ->createFusedConvActivationLayer(
            targetImpl, 1, layers[1]->getOutputTensor(0), 3, 3, 18, 8, 1, 1, 1,
            1, 1, 1, 1, 1, 1,
            "./codegen/lib/Decision_Predict/"
            "cnn_decnet0_0_Convolution_Layer_1_w.bin",
            "./codegen/lib/Decision_Predict/"
            "cnn_decnet0_0_Convolution_Layer_1_b.bin",
            0.0F, MWActivationFunctionType::ACTIVATION_FCN_ENUM::RELU, "SSCB",
            1);
    (static_cast<MWMaxPoolingLayer *>(layers[3]))
        ->createMaxPoolingLayer<float, float>(
            targetImpl, layers[2]->getOutputTensor(0), 2, 2, 2, 2, 0, 0, 0, 0,
            0, 0, "FLOAT", 1, "SSCB", 0);
    (static_cast<MWFusedConvActivationLayer *>(layers[4]))
        ->createFusedConvActivationLayer(
            targetImpl, 1, layers[3]->getOutputTensor(0), 3, 3, 8, 16, 1, 1, 1,
            1, 1, 1, 1, 1, 1,
            "./codegen/lib/Decision_Predict/cnn_decnet0_0_conv_1_w.bin",
            "./codegen/lib/Decision_Predict/cnn_decnet0_0_conv_1_b.bin", 0.0F,
            MWActivationFunctionType::ACTIVATION_FCN_ENUM::RELU, "SSCB", 1);
    (static_cast<MWMaxPoolingLayer *>(layers[5]))
        ->createMaxPoolingLayer<float, float>(
            targetImpl, layers[4]->getOutputTensor(0), 2, 2, 2, 2, 0, 0, 0, 0,
            0, 0, "FLOAT", 1, "SSCB", 0);
    (static_cast<MWFusedConvActivationLayer *>(layers[6]))
        ->createFusedConvActivationLayer(
            targetImpl, 1, layers[5]->getOutputTensor(0), 3, 3, 16, 32, 1, 1, 1,
            1, 1, 1, 1, 1, 1,
            "./codegen/lib/Decision_Predict/cnn_decnet0_0_conv_2_w.bin",
            "./codegen/lib/Decision_Predict/cnn_decnet0_0_conv_2_b.bin", 0.0F,
            MWActivationFunctionType::ACTIVATION_FCN_ENUM::RELU, "SSCB", 1);
    (static_cast<MWFCLayer *>(layers[7]))
        ->createFCLayer(targetImpl, layers[6]->getOutputTensor(0), 29760, 7,
                        "./codegen/lib/Decision_Predict/"
                        "cnn_decnet0_0_fully_Connected_Layer_w.bin",
                        "./codegen/lib/Decision_Predict/"
                        "cnn_decnet0_0_fully_Connected_Layer_b.bin",
                        "SSCB", 0);
    (static_cast<MWSoftmaxLayer *>(layers[8]))
        ->createSoftmaxLayer(targetImpl, layers[7]->getOutputTensor(0), "SSCB",
                             1);
    (static_cast<MWOutputLayer *>(layers[9]))
        ->createOutputLayer(targetImpl, layers[8]->getOutputTensor(0), "SSCB",
                            1);
    outputTensors[0] = layers[9]->getOutputTensor(0);
    setSize();
  }
  isInitialized = true;
}

//
// Arguments    : hipError_t errCode
//                const char *file
//                unsigned int b_line
// Return Type  : void
//
static void checkCleanupCudaError(hipError_t errCode, const char *file,
                                  unsigned int b_line)
{
  if ((errCode != hipSuccess) && (errCode != hipErrorDeinitialized)) {
    printf(errorString, hipGetErrorString(errCode), file, b_line);
  }
}

//
// Arguments    : const char *errMsg
//                const char *file
//                unsigned int b_line
// Return Type  : void
//
static void checkRunTimeError(const char *errMsg, const char *file,
                              unsigned int b_line)
{
  printf(errStringBase, errMsg, b_line, file);
  exit(EXIT_FAILURE);
}

//
// Arguments    : decnet0_0 *obj
// Return Type  : void
//
namespace coder {
namespace internal {
static void DeepLearningNetwork_callDelete(decnet0_0 *obj)
{
  if (obj->isInitialized) {
    obj->cleanup();
  }
}

//
// Arguments    : void
// Return Type  : ::decnet0_0
//
} // namespace internal
} // namespace coder
decnet0_0::decnet0_0()
{
  numLayers = 10;
  isInitialized = false;
  targetImpl = 0;
  layers[0] = new MWInputLayer;
  layers[0]->setName("imageinput");
  layers[1] = new MWElementwiseAffineLayer;
  layers[1]->setName("imageinput_normalization");
  layers[1]->setInPlaceIndex(0, 0);
  layers[2] = new MWFusedConvActivationLayer;
  layers[2]->setName("Convolution Layer 1_relu Layer 1");
  layers[3] = new MWMaxPoolingLayer;
  layers[3]->setName("max Pooling Layer 1");
  layers[4] = new MWFusedConvActivationLayer;
  layers[4]->setName("conv_1_relu Layer 2");
  layers[5] = new MWMaxPoolingLayer;
  layers[5]->setName("max Pooling Layer 2");
  layers[6] = new MWFusedConvActivationLayer;
  layers[6]->setName("conv_2_relu Layer 3");
  layers[7] = new MWFCLayer;
  layers[7]->setName("fully Connected Layer");
  layers[8] = new MWSoftmaxLayer;
  layers[8]->setName("softmax Layer");
  layers[9] = new MWOutputLayer;
  layers[9]->setName("classoutput");
  layers[9]->setInPlaceIndex(0, 0);
  targetImpl = new MWCudnnTarget::MWTargetNetworkImpl;
  inputTensors[0] = new MWTensor<float>;
  inputTensors[0]->setHeight(251);
  inputTensors[0]->setWidth(61);
  inputTensors[0]->setChannels(18);
  inputTensors[0]->setBatchSize(1);
  inputTensors[0]->setSequenceLength(1);
}

//
// Arguments    : void
// Return Type  : void
//
decnet0_0::~decnet0_0()
{
  try {
    if (isInitialized) {
      cleanup();
    }
    for (int idx{0}; idx < 10; idx++) {
      delete layers[idx];
    }
    if (targetImpl) {
      delete targetImpl;
    }
    delete inputTensors[0];
  } catch (...) {
  }
}

//
// Arguments    : void
// Return Type  : int
//
int decnet0_0::getBatchSize()
{
  return inputTensors[0]->getBatchSize();
}

//
// Arguments    : int b_index
// Return Type  : float *
//
float *decnet0_0::getInputDataPointer(int b_index)
{
  return (static_cast<MWTensor<float> *>(inputTensors[b_index]))->getData();
}

//
// Arguments    : void
// Return Type  : float *
//
float *decnet0_0::getInputDataPointer()
{
  return (static_cast<MWTensor<float> *>(inputTensors[0]))->getData();
}

//
// Arguments    : int layerIndex
//                int portIndex
// Return Type  : float *
//
float *decnet0_0::getLayerOutput(int layerIndex, int portIndex)
{
  return layers[layerIndex]->getLayerOutput(portIndex);
}

//
// Arguments    : int layerIndex
//                int portIndex
// Return Type  : int
//
int decnet0_0::getLayerOutputSize(int layerIndex, int portIndex)
{
  return static_cast<unsigned int>(
             layers[layerIndex]->getOutputTensor(portIndex)->getNumElements()) *
         sizeof(float);
}

//
// Arguments    : int b_index
// Return Type  : float *
//
float *decnet0_0::getOutputDataPointer(int b_index)
{
  return (static_cast<MWTensor<float> *>(outputTensors[b_index]))->getData();
}

//
// Arguments    : void
// Return Type  : float *
//
float *decnet0_0::getOutputDataPointer()
{
  return (static_cast<MWTensor<float> *>(outputTensors[0]))->getData();
}

//
// Arguments    : int layerIndex
//                int portIndex
// Return Type  : int
//
int decnet0_0::getOutputSequenceLength(int layerIndex, int portIndex)
{
  return layers[layerIndex]->getOutputTensor(portIndex)->getSequenceLength();
}

//
// Arguments    : void
// Return Type  : void
//
void decnet0_0::predict()
{
  for (int idx{0}; idx < 10; idx++) {
    layers[idx]->predict();
  }
}

//
// Arguments    : decnet0_0 *obj
// Return Type  : void
//
namespace coder {
namespace internal {
void DeepLearningNetwork_delete(decnet0_0 *obj)
{
  DeepLearningNetwork_callDelete(obj);
}

//
// Arguments    : decnet0_0 *obj
// Return Type  : void
//
void DeepLearningNetwork_setup(decnet0_0 *obj)
{
  try {
    obj->setup();
  } catch (std::runtime_error const &err) {
    obj->cleanup();
    checkRunTimeError(err.what(), __FILE__, __LINE__);
  } catch (...) {
    obj->cleanup();
    checkRunTimeError("", __FILE__, __LINE__);
  }
}

} // namespace internal
} // namespace coder

//
// File trailer for DeepLearningNetwork.cu
//
// [EOF]
//
