#include "MWCudnnCommonHeaders.hpp"
#include "MWCudnnFusedConvActivationLayerImpl.hpp"
#include <cassert>
#include <cstdio>
#include <vector>
 namespace MWCudnnTarget { 
MWFusedConvActivationLayerImpl::MWFusedConvActivationLayerImpl( MWCNNLayer* 
layer, MWTargetNetworkImpl* ntwk_impl, int filt_H, int filt_W, int numGrps, int 
numChnls, int numFilts, int FOcStuqCptsGIZXskVpC, int 
FpguQZSermqZCMRiUfML, int CTCbzQMDaLxINPbODdng, int 
CLOUhPjbgggWoXHTtmjC, int CpMjJjtGOeWOzwxpAAQP, int 
CqtPRJvHlGJFssiPzsOm, int AdmgfUbRAfzFeYHxSnQr, int 
AwZQzUhuWVLGrWgLHRuM, int enPbWLzEmxYCBmzGJutZ, const char* 
xHViLEwTujGGrPZZgmbF, const char* JxwPQNPACGfmGpNncpCY, double , 
MWActivationFunctionType::ACTIVATION_FCN_ENUM GZGFVDrXwFLJleoTDywO) : 
MWCNNLayerImpl(layer, ntwk_impl) , vIWQzNvYZSuxmOTVDFhU(NULL) , IwKnaBoXVubIRYcxEJLH(NULL) , 
TxNFOfYScyqGlEFFxbAv(NULL) , XCLDbxHBtWRStETWIkId(NULL) , 
FshVHIJMRAhtQirYPlZd(NULL) , HgeIbZCtKXtKFOEtSlPZ(NULL) , 
AzTsxYcYjIEJsGQbeYHm(filt_H) , BHuHNDGoRwGRouCxeMbw(filt_W) , 
BkwhtPQUCQKchmmimoXs(numGrps) , BUOdotSvmFyUWQKMUdra(numChnls) , 
BdqURaHPmdnfzvtUvocl(numFilts) , FLuSVNoPhAFKtLUchSvv(FOcStuqCptsGIZXskVpC) , 
FeVcBgtQmTLtmnNcJGMY(FpguQZSermqZCMRiUfML) , 
CGbFsczkgkhjcHoCKzBx(CTCbzQMDaLxINPbODdng) , 
CDJtexcMbXMWAmnNZsNf(CLOUhPjbgggWoXHTtmjC) , 
CZNYmBcNFSZWvaCklqeM(CpMjJjtGOeWOzwxpAAQP) , 
ClEhcJFlvGCgiavziIag(CqtPRJvHlGJFssiPzsOm) , 
AVeZfqOFypgpiqfRYlKc(AdmgfUbRAfzFeYHxSnQr) , 
AuqaQHxmPQSyYRemQvyX(AwZQzUhuWVLGrWgLHRuM) , 
fSbUUBgjKRbNXrHrlOLo(enPbWLzEmxYCBmzGJutZ) , 
IAlDgIFcchbwRGBSfVfA((CGbFsczkgkhjcHoCKzBx != CDJtexcMbXMWAmnNZsNf) 
|| (CZNYmBcNFSZWvaCklqeM != ClEhcJFlvGCgiavziIag)) { if 
(GZGFVDrXwFLJleoTDywO != 
MWActivationFunctionType::ACTIVATION_FCN_ENUM::RELU) { assert(false); }
#if (CUDNN_MAJOR < 6)
 throw std::runtime_error("Fused ConvReLU Layer only supported for cuDNN 6 or greater");
#else
 cRtIUoZRPICuQEOZOSzT = ntwk_impl; 
CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&NMMfJylfQjiIUAKhXCJb)); 
CUDNN_CALL(hipdnnCreateFilterDescriptor(&QMNXyOvXaZDsCpiIJPsn)); 
CUDNN_CALL(hipdnnCreateTensorDescriptor(&JsZenQeBPMhwsyEhVHiD)); 
CUDNN_CALL(hipdnnCreateActivationDescriptor(&muwRQxtWMMXAPxSuMYBw)); 
MWTensorBase* ipTensor_conv = getLayer()->getInputTensor(0); int 
NNhshzQGJHLSGjDiVerE = CGbFsczkgkhjcHoCKzBx; int 
NXruhrCCiguRjAgSNDuz = CZNYmBcNFSZWvaCklqeM; if 
(IAlDgIFcchbwRGBSfVfA) { NNhshzQGJHLSGjDiVerE = 0; 
NXruhrCCiguRjAgSNDuz = 0; TxNFOfYScyqGlEFFxbAv = new MWTensor<float>(-1, 
-1, -1, -1, -1, NULL, getLayer(), ipTensor_conv->getDataFormat(), 0); if 
(!TxNFOfYScyqGlEFFxbAv) { MWCNNLayerImpl::throwAllocationError(__LINE__, 
__FILE__); } CUDNN_CALL(hipdnnCreateTensorDescriptor(&XhAYHFyEVtlwoxGBuTpu)); } 
else { TxNFOfYScyqGlEFFxbAv = ipTensor_conv; } assert(TxNFOfYScyqGlEFFxbAv != 
NULL); bLhHPDtQpqOAnMiVledO = CGbFsczkgkhjcHoCKzBx; bQjijJlpNAVdwDDQgpaX = 
CZNYmBcNFSZWvaCklqeM; CUDNN_CALL(hipdnnSetConvolution2dDescriptor( 
NMMfJylfQjiIUAKhXCJb, NNhshzQGJHLSGjDiVerE, 
NXruhrCCiguRjAgSNDuz, FLuSVNoPhAFKtLUchSvv, FeVcBgtQmTLtmnNcJGMY, 
AVeZfqOFypgpiqfRYlKc, AuqaQHxmPQSyYRemQvyX, 
HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
#if (FP16_ENABLED == 1 && (CUDNN_MAJOR > 7 || (CUDNN_MAJOR == 7 && CUDNN_MINOR >= 2)))
 CUDNN_CALL( hipdnnSetConvolutionMathType(NMMfJylfQjiIUAKhXCJb, CUDNN_TENSOR_OP_MATH_ALLOW_CONVERSION));
#endif
 if (BkwhtPQUCQKchmmimoXs > 1) { 
CUDNN_CALL(hipdnnSetConvolutionGroupCount(NMMfJylfQjiIUAKhXCJb, 
BkwhtPQUCQKchmmimoXs)); } 
CUDNN_CALL(hipdnnSetActivationDescriptor(muwRQxtWMMXAPxSuMYBw, 
HIPDNN_ACTIVATION_RELU, HIPDNN_NOT_PROPAGATE_NAN, 0)); int 
edQOkUJIZbwzEeIcCLzG = BUOdotSvmFyUWQKMUdra * BkwhtPQUCQKchmmimoXs; 
int etjQLJVQCaeAXRWYtqOl = BdqURaHPmdnfzvtUvocl * 
BkwhtPQUCQKchmmimoXs; 
CUDNN_CALL(hipdnnSetFilter4dDescriptor(QMNXyOvXaZDsCpiIJPsn, HIPDNN_DATA_FLOAT, 
HIPDNN_TENSOR_NCHW, etjQLJVQCaeAXRWYtqOl, edQOkUJIZbwzEeIcCLzG / 
BkwhtPQUCQKchmmimoXs, AzTsxYcYjIEJsGQbeYHm, BHuHNDGoRwGRouCxeMbw)); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(JsZenQeBPMhwsyEhVHiD, HIPDNN_TENSOR_NCHW, 
HIPDNN_DATA_FLOAT, 1, etjQLJVQCaeAXRWYtqOl, 1, 1)); int weightSize = 
BUOdotSvmFyUWQKMUdra * etjQLJVQCaeAXRWYtqOl * AzTsxYcYjIEJsGQbeYHm 
* BHuHNDGoRwGRouCxeMbw; CUDA_CALL(hipMalloc((void**)&vIWQzNvYZSuxmOTVDFhU, 
sizeof(float) * weightSize)); CUDA_CALL(hipMalloc((void**)&IwKnaBoXVubIRYcxEJLH, 
sizeof(float) * etjQLJVQCaeAXRWYtqOl)); 
loadWeights(xHViLEwTujGGrPZZgmbF); loadBias(JxwPQNPACGfmGpNncpCY); createAndAddDescriptor(getLayer()->getOutputTensor(0)->getSourcePortIndex());
#endif
 } MWFusedConvActivationLayerImpl::~MWFusedConvActivationLayerImpl() { } void 
MWFusedConvActivationLayerImpl::propagateSize() {
#if (CUDNN_MAJOR >= 6)
 MWTensorBase* ipTensor_conv = getLayer()->getInputTensor(0); int inputH; int 
inputW; if (IAlDgIFcchbwRGBSfVfA) { inputH = 
ipTensor_conv->getHeight() + CGbFsczkgkhjcHoCKzBx + CDJtexcMbXMWAmnNZsNf; 
inputW = ipTensor_conv->getWidth() + CZNYmBcNFSZWvaCklqeM + 
ClEhcJFlvGCgiavziIag; } else { inputH = ipTensor_conv->getHeight(); inputW = 
ipTensor_conv->getWidth(); } TxNFOfYScyqGlEFFxbAv->setHeight(inputH); 
TxNFOfYScyqGlEFFxbAv->setWidth(inputW); 
TxNFOfYScyqGlEFFxbAv->setChannels(ipTensor_conv->getChannels()); 
TxNFOfYScyqGlEFFxbAv->setBatchSize(ipTensor_conv->getBatchSize()); 
TxNFOfYScyqGlEFFxbAv->setSequenceLength(ipTensor_conv->getSequenceLength()); 
assert(TxNFOfYScyqGlEFFxbAv->getSequenceLength() == 1); if 
(IAlDgIFcchbwRGBSfVfA) { CUDNN_CALL(hipdnnSetTensor4dDescriptor( 
XhAYHFyEVtlwoxGBuTpu, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 
TxNFOfYScyqGlEFFxbAv->getBatchSize(), TxNFOfYScyqGlEFFxbAv->getChannels(), 
TxNFOfYScyqGlEFFxbAv->getHeight(), TxNFOfYScyqGlEFFxbAv->getWidth())); } else { 
XhAYHFyEVtlwoxGBuTpu = MWCNNLayerImpl::getCuDNNDescriptor(TxNFOfYScyqGlEFFxbAv); } 
assert(BUOdotSvmFyUWQKMUdra == TxNFOfYScyqGlEFFxbAv->getChannels() / 
BkwhtPQUCQKchmmimoXs); MWTensorBase* opTensor = getLayer()->getOutputTensor(0); 
hipdnnTensorDescriptor_t* desc = getDescriptor(opTensor->getSourcePortIndex()); 
assert(desc); setDescriptor<float>(*desc, static_cast<MWTensor<float>*>(opTensor));
#if (CUDNN_MAJOR < 7)
 { CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm( 
*cRtIUoZRPICuQEOZOSzT->getCudnnHandle(), XhAYHFyEVtlwoxGBuTpu, 
QMNXyOvXaZDsCpiIJPsn, NMMfJylfQjiIUAKhXCJb, *desc, 
HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &NDjzAZSYJuWymuKDNZYB)); }
#else
 { const int maxAlgoCount(3); int returnedAlgoCount(-1); 
hipdnnConvolutionFwdAlgoPerf_t perf_results[maxAlgoCount]; 
CUDNN_CALL(cudnnGetConvolutionForwardAlgorithm_v7( 
*cRtIUoZRPICuQEOZOSzT->getCudnnHandle(), XhAYHFyEVtlwoxGBuTpu, 
QMNXyOvXaZDsCpiIJPsn, NMMfJylfQjiIUAKhXCJb, *desc, maxAlgoCount, 
&returnedAlgoCount, perf_results)); NDjzAZSYJuWymuKDNZYB = perf_results[0].algo; }
#endif
 if (CUDNN_VERSION < 7402) fixConvAlgo(); size_t sxuOMwKXOKfuExclRaSe = 0; 
CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize( 
*cRtIUoZRPICuQEOZOSzT->getCudnnHandle(), XhAYHFyEVtlwoxGBuTpu, 
QMNXyOvXaZDsCpiIJPsn, NMMfJylfQjiIUAKhXCJb, *desc, NDjzAZSYJuWymuKDNZYB, 
&sxuOMwKXOKfuExclRaSe)); if (sxuOMwKXOKfuExclRaSe > 
*cRtIUoZRPICuQEOZOSzT->getProposedWorkSpaceSize()) { 
cRtIUoZRPICuQEOZOSzT->setProposedWorkSpaceSize(sxuOMwKXOKfuExclRaSe); }
#endif
 } void MWFusedConvActivationLayerImpl::allocate() { MWTensorBase* 
ipTensor_conv = getLayer()->getInputTensor(0); if 
(IAlDgIFcchbwRGBSfVfA) { float* newInput; int inputH = 
ipTensor_conv->getHeight() + CGbFsczkgkhjcHoCKzBx + CDJtexcMbXMWAmnNZsNf; int 
inputW = ipTensor_conv->getWidth() + CZNYmBcNFSZWvaCklqeM + 
ClEhcJFlvGCgiavziIag; int paddedSize = ipTensor_conv->getBatchSize() * 
ipTensor_conv->getChannels() * inputH * inputW; 
CUDA_CALL(hipMalloc((void**)&newInput, sizeof(float) * paddedSize)); 
CUDA_CALL(hipMemset(newInput, 0, sizeof(float) * paddedSize)); 
static_cast<MWTensor<float>*>(TxNFOfYScyqGlEFFxbAv)->setData(newInput); } 
XCLDbxHBtWRStETWIkId = 
static_cast<MWTensor<float>*>(getLayer()->getOutputTensor(0))->getData(); 
setalpha2Ptr(getZeroPtr()); int numInputs = getLayer()->getNumInputs(); if 
(numInputs == 2) { setalpha2Ptr(getOnePtr()); XCLDbxHBtWRStETWIkId = 
static_cast<MWTensor<float>*>(getLayer()->getInputTensor(1))->getData(); } if 
(static_cast<MWTensor<float>*>(TxNFOfYScyqGlEFFxbAv)->getData() == 
XCLDbxHBtWRStETWIkId) { int xInputTensorSize = 
getLayer()->getInputTensor(0)->getNumElements(); CUDA_CALL( 
hipMalloc((void**)&FshVHIJMRAhtQirYPlZd, sizeof(float) * 
xInputTensorSize)); } } void MWFusedConvActivationLayerImpl::deallocate() { if 
(TxNFOfYScyqGlEFFxbAv != getLayer()->getInputTensor(0)) { 
assert(IAlDgIFcchbwRGBSfVfA); 
CUDA_FREE_CALL(static_cast<MWTensor<float>*>(TxNFOfYScyqGlEFFxbAv)->getData()); 
static_cast<MWTensor<float>*>(TxNFOfYScyqGlEFFxbAv)->setData((float*)NULL); } if 
(FshVHIJMRAhtQirYPlZd) { CUDA_FREE_CALL(FshVHIJMRAhtQirYPlZd); 
FshVHIJMRAhtQirYPlZd = NULL; } } void 
MWFusedConvActivationLayerImpl::predict() { MWCNNLayer* fusedConvReluLayer = 
getLayer(); MWTensorBase* ipTensorBase = fusedConvReluLayer->getInputTensor(); 
MWTensorBase* opTensorBase = fusedConvReluLayer->getOutputTensor(); 
MWTensor<float>* ipTensor = static_cast<MWTensor<float>*>(ipTensorBase); 
MWTensor<float>* opTensor = static_cast<MWTensor<float>*>(opTensorBase); if 
(TxNFOfYScyqGlEFFxbAv != fusedConvReluLayer->getInputTensor()) { 
CUDA_CALL(hipMemset(static_cast<MWTensor<float>*>(TxNFOfYScyqGlEFFxbAv)->getData(), 
0, sizeof(float) * TxNFOfYScyqGlEFFxbAv->getNumElements())); 
MWCNNLayerImpl::padInput(ipTensor->getData(), ipTensor->getHeight(), 
ipTensor->getWidth(), ipTensor->getChannels(), TxNFOfYScyqGlEFFxbAv->getHeight(), 
TxNFOfYScyqGlEFFxbAv->getWidth(), bLhHPDtQpqOAnMiVledO, bQjijJlpNAVdwDDQgpaX, 
static_cast<MWTensor<float>*>(TxNFOfYScyqGlEFFxbAv)->getData(), 
ipTensor->getNumElements()); } hipdnnTensorDescriptor_t* desc = 
getDescriptor(opTensor->getSourcePortIndex()); assert(desc);
#if (CUDNN_MAJOR >= 6)
 assert(opTensor->getData() != 
static_cast<MWTensor<float>*>(TxNFOfYScyqGlEFFxbAv)->getData() || 
(getLayer()->getNumInputs() == 2)); float* pzUAoBDvaKAtdsmkQuct; if 
(static_cast<MWTensor<float>*>(TxNFOfYScyqGlEFFxbAv)->getData() == 
XCLDbxHBtWRStETWIkId) { CUDA_CALL(hipMemcpy(FshVHIJMRAhtQirYPlZd, 
static_cast<MWTensor<float>*>(TxNFOfYScyqGlEFFxbAv)->getData(), sizeof(float) * 
opTensorBase->getNumElements(), hipMemcpyDeviceToDevice)); 
pzUAoBDvaKAtdsmkQuct = FshVHIJMRAhtQirYPlZd; } else { 
pzUAoBDvaKAtdsmkQuct = 
static_cast<MWTensor<float>*>(TxNFOfYScyqGlEFFxbAv)->getData(); } 
CUDNN_CALL(cudnnConvolutionBiasActivationForward( 
*cRtIUoZRPICuQEOZOSzT->getCudnnHandle(), getOnePtr(), XhAYHFyEVtlwoxGBuTpu, 
pzUAoBDvaKAtdsmkQuct, QMNXyOvXaZDsCpiIJPsn, vIWQzNvYZSuxmOTVDFhU, 
NMMfJylfQjiIUAKhXCJb, NDjzAZSYJuWymuKDNZYB, cRtIUoZRPICuQEOZOSzT->getWorkSpace(), 
*cRtIUoZRPICuQEOZOSzT->getAllocatedWorkSpaceSize(), getalpha2Ptr(),  *desc,  
XCLDbxHBtWRStETWIkId,  JsZenQeBPMhwsyEhVHiD, IwKnaBoXVubIRYcxEJLH, muwRQxtWMMXAPxSuMYBw, 
*desc, opTensor->getData()));
#endif
 } void MWFusedConvActivationLayerImpl::cleanup() { 
CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(NMMfJylfQjiIUAKhXCJb)); 
CUDNN_CALL(hipdnnDestroyFilterDescriptor(QMNXyOvXaZDsCpiIJPsn)); 
CUDNN_CALL(hipdnnDestroyActivationDescriptor(muwRQxtWMMXAPxSuMYBw)); if 
(vIWQzNvYZSuxmOTVDFhU) { CUDA_FREE_CALL(vIWQzNvYZSuxmOTVDFhU); vIWQzNvYZSuxmOTVDFhU = NULL; } 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(JsZenQeBPMhwsyEhVHiD)); if 
(IwKnaBoXVubIRYcxEJLH) { CUDA_FREE_CALL(IwKnaBoXVubIRYcxEJLH); IwKnaBoXVubIRYcxEJLH = NULL; } if 
(TxNFOfYScyqGlEFFxbAv != getLayer()->getInputTensor(0)) { 
assert(IAlDgIFcchbwRGBSfVfA); 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(XhAYHFyEVtlwoxGBuTpu)); } } void 
MWFusedConvActivationLayerImpl::loadWeights(const char* PtRNGuserCxHAQfyEjFc) { 
FILE* QTXuPiGKeBUnmRzhlIDp = MWCNNLayer::openBinaryFile(PtRNGuserCxHAQfyEjFc); 
assert(QTXuPiGKeBUnmRzhlIDp); int cnEykmOGhLuyKuadExWe = BUOdotSvmFyUWQKMUdra * 
BkwhtPQUCQKchmmimoXs * BdqURaHPmdnfzvtUvocl * AzTsxYcYjIEJsGQbeYHm * 
BHuHNDGoRwGRouCxeMbw;  float* KZWeXiYFmdpQdsgidKeG = MALLOC_CALL(sizeof(float) * 
cnEykmOGhLuyKuadExWe); MWCNNLayer::call_fread(KZWeXiYFmdpQdsgidKeG, sizeof(float), 
cnEykmOGhLuyKuadExWe, QTXuPiGKeBUnmRzhlIDp, PtRNGuserCxHAQfyEjFc); 
CUDA_CALL(hipMemcpy(vIWQzNvYZSuxmOTVDFhU, KZWeXiYFmdpQdsgidKeG, sizeof(float) * 
cnEykmOGhLuyKuadExWe, hipMemcpyHostToDevice));
#if 0
 printf("%s loaded. Size = %d. %f\n", PtRNGuserCxHAQfyEjFc, cnEykmOGhLuyKuadExWe, KZWeXiYFmdpQdsgidKeG[0]);
#endif
 free(KZWeXiYFmdpQdsgidKeG); fclose(QTXuPiGKeBUnmRzhlIDp); return; } void 
MWFusedConvActivationLayerImpl::loadBias(const char* PtRNGuserCxHAQfyEjFc) { 
FILE* QTXuPiGKeBUnmRzhlIDp = MWCNNLayer::openBinaryFile(PtRNGuserCxHAQfyEjFc); 
assert(QTXuPiGKeBUnmRzhlIDp); int cnEykmOGhLuyKuadExWe = BkwhtPQUCQKchmmimoXs * 
BdqURaHPmdnfzvtUvocl;  float* KZWeXiYFmdpQdsgidKeG = MALLOC_CALL(sizeof(float) * 
cnEykmOGhLuyKuadExWe); MWCNNLayer::call_fread(KZWeXiYFmdpQdsgidKeG, sizeof(float), 
cnEykmOGhLuyKuadExWe, QTXuPiGKeBUnmRzhlIDp, PtRNGuserCxHAQfyEjFc); 
CUDA_CALL(hipMemcpy(IwKnaBoXVubIRYcxEJLH, KZWeXiYFmdpQdsgidKeG, sizeof(float) * 
cnEykmOGhLuyKuadExWe, hipMemcpyHostToDevice)); free(KZWeXiYFmdpQdsgidKeG); 
fclose(QTXuPiGKeBUnmRzhlIDp); return; } void 
MWFusedConvActivationLayerImpl::setLearnables(std::vector<float*> learnables) { 
assert(learnables.size() == 2); int cnEykmOGhLuyKuadExWe = BUOdotSvmFyUWQKMUdra * 
BdqURaHPmdnfzvtUvocl * BkwhtPQUCQKchmmimoXs * AzTsxYcYjIEJsGQbeYHm * 
BHuHNDGoRwGRouCxeMbw;  float* wqggPBXZvtlxnxwngvAq = learnables[0]; 
CUDA_CALL(hipMemcpy(vIWQzNvYZSuxmOTVDFhU, wqggPBXZvtlxnxwngvAq, sizeof(float) * 
cnEykmOGhLuyKuadExWe, hipMemcpyHostToDevice)); cnEykmOGhLuyKuadExWe = 
BdqURaHPmdnfzvtUvocl * BkwhtPQUCQKchmmimoXs;  float* JgLfgHrHMEMmMYTettJF = 
learnables[1]; CUDA_CALL(hipMemcpy(IwKnaBoXVubIRYcxEJLH, JgLfgHrHMEMmMYTettJF, 
sizeof(float) * cnEykmOGhLuyKuadExWe, hipMemcpyHostToDevice)); } void 
MWFusedConvActivationLayerImpl::postSetup() { if 
(cRtIUoZRPICuQEOZOSzT->getAutoTune()) { getConvAlgoTuned(); } else { 
getConvAlgoWorkSpaceLimit(); } } void 
MWFusedConvActivationLayerImpl::getConvAlgoTuned() { MWTensorBase* opTensorBase 
= getLayer()->getOutputTensor(0); MWTensor<float>* opTensor = 
static_cast<MWTensor<float>*>(opTensorBase); hipdnnConvolutionFwdAlgoPerf_t 
perf_results[3]; hipdnnTensorDescriptor_t* desc = 
getDescriptor(getLayer()->getOutputTensor()->getSourcePortIndex()); 
assert(desc); int returnedAlgoCount; 
CUDNN_CALL(hipdnnFindConvolutionForwardAlgorithmEx( 
*cRtIUoZRPICuQEOZOSzT->getCudnnHandle(), XhAYHFyEVtlwoxGBuTpu, 
static_cast<MWTensor<float>*>(TxNFOfYScyqGlEFFxbAv)->getData(), 
QMNXyOvXaZDsCpiIJPsn, vIWQzNvYZSuxmOTVDFhU, NMMfJylfQjiIUAKhXCJb, *desc, 
opTensor->getData(), 3, &returnedAlgoCount, &perf_results[0], 
cRtIUoZRPICuQEOZOSzT->getWorkSpace(), 
*cRtIUoZRPICuQEOZOSzT->getAllocatedWorkSpaceSize())); NDjzAZSYJuWymuKDNZYB = 
perf_results[0].algo; if (CUDNN_VERSION < 7402) fixConvAlgo(); } void 
MWFusedConvActivationLayerImpl::getConvAlgoWorkSpaceLimit() { 
hipdnnTensorDescriptor_t* desc = 
getDescriptor(getLayer()->getOutputTensor()->getSourcePortIndex()); assert(desc);
#if (CUDNN_MAJOR < 8)
 CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm( 
*cRtIUoZRPICuQEOZOSzT->getCudnnHandle(), XhAYHFyEVtlwoxGBuTpu, 
QMNXyOvXaZDsCpiIJPsn, NMMfJylfQjiIUAKhXCJb, *desc, 
HIPDNN_CONVOLUTION_FWD_SPECIFY_WORKSPACE_LIMIT, 
*cRtIUoZRPICuQEOZOSzT->getAllocatedWorkSpaceSize(), &NDjzAZSYJuWymuKDNZYB));
#else
 int maxAlgoCount(-1); 
CUDNN_CALL(cudnnGetConvolutionForwardAlgorithmMaxCount(*cRtIUoZRPICuQEOZOSzT->getCudnnHandle(), 
&maxAlgoCount)); int returnedAlgoCount(-1); 
std::vector<hipdnnConvolutionFwdAlgoPerf_t> perf_results(maxAlgoCount); 
CUDNN_CALL(cudnnGetConvolutionForwardAlgorithm_v7( 
*cRtIUoZRPICuQEOZOSzT->getCudnnHandle(), XhAYHFyEVtlwoxGBuTpu, 
QMNXyOvXaZDsCpiIJPsn, NMMfJylfQjiIUAKhXCJb, *desc, maxAlgoCount, 
&returnedAlgoCount, &perf_results[0])); hipdnnConvolutionFwdAlgoPerf_t 
nextFastest; bool algoFound(false); for (int i = 0; i < returnedAlgoCount; ++i) 
{ nextFastest = perf_results[i]; if (nextFastest.memory <= 
*cRtIUoZRPICuQEOZOSzT->getAllocatedWorkSpaceSize()) { NDjzAZSYJuWymuKDNZYB = 
nextFastest.algo; algoFound = true; break; } } if (!algoFound) { assert(false); }
#endif
 if (CUDNN_VERSION < 7402) fixConvAlgo(); } void 
MWFusedConvActivationLayerImpl::fixConvAlgo() { int inputH = 
TxNFOfYScyqGlEFFxbAv->getHeight(); int inputW = TxNFOfYScyqGlEFFxbAv->getWidth(); 
if (NDjzAZSYJuWymuKDNZYB == HIPDNN_CONVOLUTION_FWD_ALGO_FFT && (inputH > 64 || 
inputW > 64)) { NDjzAZSYJuWymuKDNZYB = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM; 
} } } 