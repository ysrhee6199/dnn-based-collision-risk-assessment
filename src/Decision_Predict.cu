//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: Decision_Predict.cu
//
// GPU Coder version                    : 2.5
// CUDA/C/C++ source code generated on  : 11-Jun-2024 01:16:03
//

// Include Files
#include "Decision_Predict.h"
#include "Decision_Predict_data.h"
#include "Decision_Predict_initialize.h"
#include "Decision_Predict_internal_types.h"
#include "DeepLearningNetwork.h"
#include "predict.h"

// Variable Definitions
static decnet0_0 DecNet;

static bool DecNet_not_empty;

// Function Definitions
//
// Arguments    : const unsigned char cpu_in[275598]
//                float cpu_out[7]
// Return Type  : void
//
void Decision_Predict(const unsigned char cpu_in[275598], float cpu_out[7])
{
  float(*gpu_out)[7];
  unsigned char(*gpu_in)[275598];
  if (!isInitialized_Decision_Predict) {
    Decision_Predict_initialize();
  }
  hipMalloc(&gpu_out, 28UL);
  hipMalloc(&gpu_in, 275598UL);
  if (!DecNet_not_empty) {
    coder::internal::DeepLearningNetwork_setup(&DecNet);
    DecNet.matlabCodegenIsDeleted = false;
    DecNet_not_empty = true;
  }
  hipMemcpy(*gpu_in, cpu_in, 275598UL, hipMemcpyHostToDevice);
  coder::internal::DeepLearningNetwork_predict(&DecNet, *gpu_in, *gpu_out);
  hipMemcpy(cpu_out, *gpu_out, 28UL, hipMemcpyDeviceToHost);
  hipFree(*gpu_in);
  hipFree(*gpu_out);
}

//
// Arguments    : void
// Return Type  : void
//
void Decision_Predict_free()
{
  if (!DecNet.matlabCodegenIsDeleted) {
    DecNet.matlabCodegenIsDeleted = true;
    coder::internal::DeepLearningNetwork_delete(&DecNet);
  }
}

//
// Arguments    : void
// Return Type  : void
//
void Decision_Predict_init()
{
  DecNet_not_empty = false;
  DecNet.matlabCodegenIsDeleted = true;
}

//
// File trailer for Decision_Predict.cu
//
// [EOF]
//
