#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: predict.cu
//
// GPU Coder version                    : 2.5
// CUDA/C/C++ source code generated on  : 11-Jun-2024 01:16:03
//

// Include Files
#include "predict.h"
#include "Decision_Predict_internal_types.h"
#include "DeepLearningNetwork.h"
#include "MWCudaDimUtility.hpp"

// Function Declarations
static __global__ void
DeepLearningNetwork_predict_kernel1(const unsigned char varargin_1[275598],
                                    float inMiniBatchGroup_0_f1[275598]);

// Function Definitions
//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned char varargin_1[275598]
//                float inMiniBatchGroup_0_f1[275598]
// Return Type  : void
//
static __global__
    __launch_bounds__(512, 1) void DeepLearningNetwork_predict_kernel1(
        const unsigned char varargin_1[275598],
        float inMiniBatchGroup_0_f1[275598])
{
  unsigned long threadId;
  int i;
  int i1;
  int p;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  i = static_cast<int>(threadId % 61UL);
  threadId = (threadId - static_cast<unsigned long>(i)) / 61UL;
  i1 = static_cast<int>(threadId % 251UL);
  threadId = (threadId - static_cast<unsigned long>(i1)) / 251UL;
  p = static_cast<int>(threadId);
  if ((p < 18) && (i1 < 251) && (i < 61)) {
    inMiniBatchGroup_0_f1[(i + 61 * i1) + 15311 * p] =
        static_cast<float>(varargin_1[(i1 + 251 * i) + 15311 * p]);
  }
}

//
// Arguments    : decnet0_0 *obj
//                const unsigned char varargin_1[275598]
//                float varargout_1[7]
// Return Type  : void
//
namespace coder {
namespace internal {
void DeepLearningNetwork_predict(decnet0_0 *obj,
                                 const unsigned char varargin_1[275598],
                                 float varargout_1[7])
{
  float(*gpu_inMiniBatchGroup_0_f1)[275598];
  hipMalloc(&gpu_inMiniBatchGroup_0_f1, 1102392UL);
  DeepLearningNetwork_predict_kernel1<<<dim3(539U, 1U, 1U),
                                        dim3(512U, 1U, 1U)>>>(
      varargin_1, *gpu_inMiniBatchGroup_0_f1);
  hipMemcpy(obj->getInputDataPointer(0), *gpu_inMiniBatchGroup_0_f1,
             obj->getLayerOutputSize(0, 0), hipMemcpyDeviceToDevice);
  obj->predict();
  hipMemcpy(varargout_1, obj->getLayerOutput(9, 0),
             obj->getLayerOutputSize(9, 0), hipMemcpyDeviceToDevice);
  hipFree(*gpu_inMiniBatchGroup_0_f1);
}

} // namespace internal
} // namespace coder

//
// File trailer for predict.cu
//
// [EOF]
//
