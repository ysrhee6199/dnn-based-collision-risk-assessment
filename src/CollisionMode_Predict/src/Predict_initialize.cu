//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: Predict_initialize.cu
//
// GPU Coder version                    : 2.5
// CUDA/C/C++ source code generated on  : 31-Jul-2024 04:58:22
//

// Include Files
#include "Predict_initialize.h"
#include "Predict.h"
#include "Predict_data.h"

// Function Definitions
//
// Arguments    : void
// Return Type  : void
//
void Predict_initialize()
{
  Predict_init();
  hipGetLastError();
  isInitialized_Predict = true;
}

//
// File trailer for Predict_initialize.cu
//
// [EOF]
//
