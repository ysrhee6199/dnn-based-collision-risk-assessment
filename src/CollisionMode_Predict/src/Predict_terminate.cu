//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: Predict_terminate.cu
//
// GPU Coder version                    : 2.5
// CUDA/C/C++ source code generated on  : 31-Jul-2024 04:58:22
//

// Include Files
#include "Predict_terminate.h"
#include "Predict.h"
#include "Predict_data.h"
#include "stdio.h"

// Function Definitions
//
// Arguments    : void
// Return Type  : void
//
void Predict_terminate()
{
  hipError_t errCode;
  errCode = hipGetLastError();
  if (errCode != hipSuccess) {
    fprintf(stderr, "ERR[%d] %s:%s\n", errCode, hipGetErrorName(errCode),
            hipGetErrorString(errCode));
    exit(errCode);
  }
  Predict_free();
  isInitialized_Predict = false;
}

//
// File trailer for Predict_terminate.cu
//
// [EOF]
//
