//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: Predict.cu
//
// GPU Coder version                    : 2.5
// CUDA/C/C++ source code generated on  : 31-Jul-2024 04:58:22
//

// Include Files
#include "Predict.h"
#include "DeepLearningNetwork.h"
#include "Predict_data.h"
#include "Predict_initialize.h"
#include "Predict_internal_types.h"
#include "predict1.h"

// Variable Definitions
static decnet0_0 DecNet;

static bool DecNet_not_empty;

// Function Definitions
//
// Arguments    : const unsigned char cpu_in[60903]
//                float cpu_out[13]
// Return Type  : void
//
void Predict(const unsigned char cpu_in[60903], float cpu_out[13])
{
  float(*gpu_out)[13];
  unsigned char(*gpu_in)[60903];
  if (!isInitialized_Predict) {
    Predict_initialize();
  }
  hipMalloc(&gpu_out, 52UL);
  hipMalloc(&gpu_in, 60903UL);
  if (!DecNet_not_empty) {
    coder::internal::DeepLearningNetwork_setup(&DecNet);
    DecNet.matlabCodegenIsDeleted = false;
    DecNet_not_empty = true;
  }
  hipMemcpy(*gpu_in, cpu_in, 60903UL, hipMemcpyHostToDevice);
  coder::internal::DeepLearningNetwork_predict(&DecNet, *gpu_in, *gpu_out);
  hipMemcpy(cpu_out, *gpu_out, 52UL, hipMemcpyDeviceToHost);
  hipFree(*gpu_in);
  hipFree(*gpu_out);
}

//
// Arguments    : void
// Return Type  : void
//
void Predict_free()
{
  if (!DecNet.matlabCodegenIsDeleted) {
    DecNet.matlabCodegenIsDeleted = true;
    coder::internal::DeepLearningNetwork_delete(&DecNet);
  }
}

//
// Arguments    : void
// Return Type  : void
//
void Predict_init()
{
  DecNet_not_empty = false;
  DecNet.matlabCodegenIsDeleted = true;
}

//
// File trailer for Predict.cu
//
// [EOF]
//
