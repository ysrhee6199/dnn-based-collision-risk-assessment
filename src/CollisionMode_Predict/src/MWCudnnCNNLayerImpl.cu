#include "hip/hip_runtime.h"
#include "MWCudnnCommonHeaders.hpp"
#include <cstdlib>
#include <cassert>
#include <cstdio>
#include <stdexcept>
#include <sstream>
 namespace MWCudnnTarget { float* malloc_call_line_file(size_t msize, const int 
aFDPITUhkPdupMfPOBnd, const char* PmFfARVzoHVAYkfpuvqK) { float* mem = 
(float*)malloc(msize); if (!mem) { 
MWCNNLayerImpl::throwAllocationError(aFDPITUhkPdupMfPOBnd, PmFfARVzoHVAYkfpuvqK); } 
return mem; } void cuda_call_line_file(hipError_t rMMjgjGRAiLVlTlRSByU, const int 
aFDPITUhkPdupMfPOBnd, const char* PmFfARVzoHVAYkfpuvqK) { if (rMMjgjGRAiLVlTlRSByU != 
hipSuccess) { throw_cuda_error(rMMjgjGRAiLVlTlRSByU, aFDPITUhkPdupMfPOBnd, 
PmFfARVzoHVAYkfpuvqK); } } void throw_cuda_error(hipError_t rMMjgjGRAiLVlTlRSByU, 
const int aFDPITUhkPdupMfPOBnd, const char* PmFfARVzoHVAYkfpuvqK) { std::stringstream ss; 
ss << "CUDA Error " << rMMjgjGRAiLVlTlRSByU << "(" << 
hipGetErrorString(rMMjgjGRAiLVlTlRSByU) << ") at line " << aFDPITUhkPdupMfPOBnd << 
", file: " << PmFfARVzoHVAYkfpuvqK << "\n"; rMMjgjGRAiLVlTlRSByU = hipGetLastError();  
throw std::runtime_error(ss.str()); } void cudnn_call_line_file(hipdnnStatus_t 
rMMjgjGRAiLVlTlRSByU, const int aFDPITUhkPdupMfPOBnd, const char* PmFfARVzoHVAYkfpuvqK) { if 
(rMMjgjGRAiLVlTlRSByU != HIPDNN_STATUS_SUCCESS) { std::stringstream ss; ss << 
"CuDNN Error " << rMMjgjGRAiLVlTlRSByU << "(" << 
hipdnnGetErrorString(rMMjgjGRAiLVlTlRSByU) << ") at line " << aFDPITUhkPdupMfPOBnd << 
", file: " << PmFfARVzoHVAYkfpuvqK << "\n"; throw std::runtime_error(ss.str()); } } 
const char* cublasGetErrorString(hipblasStatus_t rMMjgjGRAiLVlTlRSByU) { switch 
(rMMjgjGRAiLVlTlRSByU) { case HIPBLAS_STATUS_SUCCESS: return 
"HIPBLAS_STATUS_SUCCESS"; case HIPBLAS_STATUS_NOT_INITIALIZED: return 
"HIPBLAS_STATUS_NOT_INITIALIZED"; case HIPBLAS_STATUS_ALLOC_FAILED: return 
"HIPBLAS_STATUS_ALLOC_FAILED"; case HIPBLAS_STATUS_INVALID_VALUE: return 
"HIPBLAS_STATUS_INVALID_VALUE"; case HIPBLAS_STATUS_ARCH_MISMATCH: return 
"HIPBLAS_STATUS_ARCH_MISMATCH"; case HIPBLAS_STATUS_MAPPING_ERROR: return 
"HIPBLAS_STATUS_MAPPING_ERROR"; case HIPBLAS_STATUS_EXECUTION_FAILED: return 
"HIPBLAS_STATUS_EXECUTION_FAILED"; case HIPBLAS_STATUS_INTERNAL_ERROR: return 
"HIPBLAS_STATUS_INTERNAL_ERROR"; case HIPBLAS_STATUS_NOT_SUPPORTED: return 
"HIPBLAS_STATUS_NOT_SUPPORTED"; case HIPBLAS_STATUS_UNKNOWN: return 
"HIPBLAS_STATUS_UNKNOWN"; } return "unknown error"; } void 
cublas_call_line_file(hipblasStatus_t rMMjgjGRAiLVlTlRSByU, const int 
aFDPITUhkPdupMfPOBnd, const char* PmFfARVzoHVAYkfpuvqK) { if (rMMjgjGRAiLVlTlRSByU != 
HIPBLAS_STATUS_SUCCESS) { std::stringstream ss; ss << "CuBLAS Error " << 
rMMjgjGRAiLVlTlRSByU << "(" << cublasGetErrorString(rMMjgjGRAiLVlTlRSByU) << 
") at line " << aFDPITUhkPdupMfPOBnd << ", file: " << PmFfARVzoHVAYkfpuvqK << "\n"; throw 
std::runtime_error(ss.str()); } } MWCNNLayerImpl::MWCNNLayerImpl(MWCNNLayer* 
layer, MWTargetNetworkImpl* ntwk_impl) : MWCNNLayerImplBase(layer) , 
OzygUJRIZYnGLzSjgahB(0.0) , OwenhowBxTAXHXmJpIKd(1.0) , OiVqrkNdXioJhALWMMvm(-1.0) , 
cRtIUoZRPICuQEOZOSzT(ntwk_impl) { } MWCNNLayerImpl::~MWCNNLayerImpl() { for 
(std::map<int, hipdnnTensorDescriptor_t*>::iterator it = 
jmcFOAbZArjGDNhshSro.begin(); it != jmcFOAbZArjGDNhshSro.end(); ++it) { 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*it->second)); delete it->second; 
it->second = 0; } } void MWCNNLayerImpl::allocateOutput(int outIdx) { if 
(getLayer()->getOutputTensor(outIdx)->isFloat()) { 
this->allocateOutputImpl<float>(outIdx); } else { 
assert(getLayer()->getOutputTensor(outIdx)->isInt8()); 
this->allocateOutputImpl<signed char>(outIdx); } } template <class T> void 
MWCNNLayerImpl::allocateOutputImpl(int outIdx) { MWTensorBase* opTensorBase = 
getLayer()->getOutputTensor(outIdx); bool bufferReuse = 
opTensorBase->getopBufIndex() >= 0; if (bufferReuse) { 
assert(opTensorBase->isFloat()); MWTensor<float>* opTensor = 
static_cast<MWTensor<float>*>(opTensorBase); float* buffer = 
cRtIUoZRPICuQEOZOSzT->getBufferPtr(opTensor->getopBufIndex()); int inIdx = 
getLayer()->getInPlaceIndex(outIdx); if (inIdx != -1) { MWTensor<float>* 
ipTensor = static_cast<MWTensor<float>*>(getLayer()->getInputTensor(inIdx)); 
buffer = ipTensor->getData(); assert(buffer); } float* opPointer = 
offsetOutputPointer(buffer, outIdx); assert(opPointer); 
opTensor->setData(opPointer); } else { int inIdx = 
getLayer()->getInPlaceIndex(outIdx); if (inIdx != -1) { MWTensor<T>* ipTensor = 
static_cast<MWTensor<T>*>(getLayer()->getInputTensor(inIdx)); MWTensor<T>* 
opTensor = static_cast<MWTensor<T>*>(opTensorBase); T* ipData = 
ipTensor->getData(); assert(ipData); T* opPointer = offsetOutputPointer(ipData, 
outIdx); assert(opPointer); opTensor->setData(opPointer); } else { MWTensor<T>* 
opTensor = static_cast<MWTensor<T>*>(opTensorBase); T* OAKPrVDonUthXHZkRzEc; 
CUDA_CALL(hipMalloc((void**)&OAKPrVDonUthXHZkRzEc, sizeof(T) * 
opTensor->getNumElements())); opTensor->setData(OAKPrVDonUthXHZkRzEc); } } } 
template void MWCNNLayerImpl::allocateOutputImpl<float>(int); template void 
MWCNNLayerImpl::allocateOutputImpl<signed char>(int); void 
MWCNNLayerImpl::deallocateOutput(int outIdx) { if 
(getLayer()->getOutputTensor(outIdx)->isFloat()) { 
this->deallocateOutputImpl<float>(outIdx); } else { 
assert(getLayer()->getOutputTensor(outIdx)->isInt8()); 
this->deallocateOutputImpl<signed char>(outIdx); } } template <class T> void 
MWCNNLayerImpl::deallocateOutputImpl(int outIdx) { if 
(getLayer()->getInPlaceIndex(outIdx) == -1) { MWTensor<T>* opTensor = 
static_cast<MWTensor<T>*>(getLayer()->getOutputTensor(outIdx)); T* data = 
opTensor->getData(); CUDA_FREE_CALL(data); } } template void 
MWCNNLayerImpl::deallocateOutputImpl<float>(int); template void 
MWCNNLayerImpl::deallocateOutputImpl<signed char>(int); float* 
MWCNNLayerImpl::getZeroPtr() { return &OzygUJRIZYnGLzSjgahB; } float* 
MWCNNLayerImpl::getOnePtr() { return &OwenhowBxTAXHXmJpIKd; } float* 
MWCNNLayerImpl::getNegOnePtr() { return &OiVqrkNdXioJhALWMMvm; } 
hipdnnTensorDescriptor_t* MWCNNLayerImpl::createAndAddDescriptor(int index) { 
std::map<int, hipdnnTensorDescriptor_t*>::iterator it = 
jmcFOAbZArjGDNhshSro.find(index); assert(it == jmcFOAbZArjGDNhshSro.end()); 
hipdnnTensorDescriptor_t* newDescriptor = new hipdnnTensorDescriptor_t; if 
(!newDescriptor) { MWCNNLayerImpl::throwAllocationError(__LINE__, __FILE__); } 
jmcFOAbZArjGDNhshSro[index] = newDescriptor; 
CUDNN_CALL(hipdnnCreateTensorDescriptor(newDescriptor)); return newDescriptor; } 
hipdnnTensorDescriptor_t* MWCNNLayerImpl::getDescriptor(int index) { 
std::map<int, hipdnnTensorDescriptor_t*>::iterator it = 
jmcFOAbZArjGDNhshSro.find(index); if (it != jmcFOAbZArjGDNhshSro.end()) { 
return it->second; } else { return NULL; } } template <class T> void 
MWCNNLayerImpl::setDescriptor(hipdnnTensorDescriptor_t& desc, MWTensor<T>* 
tensor) { if (tensor->getSequenceLength() == 1) { 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, 
MWCNNLayerImpl::getCuDNNDataType<T>(), tensor->getBatchSize(),  
tensor->getChannels(),  tensor->getHeight(),  tensor->getWidth()));  } else { 
int dims[5] = {tensor->getSequenceLength(), tensor->getBatchSize(), 
tensor->getChannels(), tensor->getHeight(), tensor->getWidth()}; int 
strides[5]; MWTensorBase::getStrides(dims, 5, strides); 
CUDNN_CALL(hipdnnSetTensorNdDescriptor(desc, 
MWCNNLayerImpl::getCuDNNDataType<T>(), 5, dims, strides)); } } template void 
MWCNNLayerImpl::setDescriptor<float>(hipdnnTensorDescriptor_t&, 
MWTensor<float>*); template void MWCNNLayerImpl::setDescriptor<signed 
char>(hipdnnTensorDescriptor_t&, MWTensor<signed char>*); template <> 
hipdnnDataType_t MWCNNLayerImpl::getCuDNNDataType<float>() { return 
HIPDNN_DATA_FLOAT; } template <> hipdnnDataType_t 
MWCNNLayerImpl::getCuDNNDataType<signed char>() { return HIPDNN_DATA_INT8; } 
hipdnnTensorDescriptor_t MWCNNLayerImpl::getCuDNNDescriptor(MWTensorBase* 
tensor) { MWCNNLayer* layer = tensor->getOwner(); MWCNNLayerImpl* impl = 
static_cast<MWCNNLayerImpl*>(layer->getImpl()); if (impl) { 
hipdnnTensorDescriptor_t* desc = 
impl->getDescriptor(tensor->getSourcePortIndex()); if (desc == NULL) { 
impl->createAndAddDescriptor(tensor->getSourcePortIndex()); desc = 
impl->getDescriptor(tensor->getSourcePortIndex()); assert(desc); } if 
(tensor->isFloat()) { MWCNNLayerImpl::setDescriptor<float>(*desc, 
static_cast<MWTensor<float>*>(tensor)); } else { assert(tensor->isInt8()); 
MWCNNLayerImpl::setDescriptor<signed char>(*desc, static_cast<MWTensor<signed 
char>*>(tensor)); } return *desc; } else { hipdnnTensorDescriptor_t 
tmpDescriptor; CUDNN_CALL(hipdnnCreateTensorDescriptor(&tmpDescriptor)); if 
(tensor->isFloat()) { MWCNNLayerImpl::setDescriptor<float>(tmpDescriptor, 
static_cast<MWTensor<float>*>(tensor)); } else { assert(tensor->isInt8()); 
MWCNNLayerImpl::setDescriptor<signed char>(tmpDescriptor, 
static_cast<MWTensor<signed char>*>(tensor)); } return tmpDescriptor; } } void 
__global__ __launch_bounds__(1024) padInputImpl(float* in, int inputH, int 
inputW, int inputCh, int outputH, int outputW, int offsetH, int offsetW, float* 
out, int inputElems) { for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < 
inputElems; i += blockDim.x * gridDim.x) { int idxB = i / (inputH * inputW * 
inputCh); int rem = (i - idxB * (inputH * inputW * inputCh)); int idxCh = rem / 
(inputH * inputW); int rem1 = rem - idxCh * (inputH * inputW); int idxH = rem1 
/ inputW; int idxCol = rem1 - idxH * inputW; if ((idxH < inputH) && (idxCol < 
inputW)) { int outputR = idxH + offsetH; int outputCol = idxCol + offsetW; int 
outputCh = inputCh; out[idxB * (outputH * outputW * outputCh) + idxCh * 
(outputH * outputW) + outputR * (outputW) + outputCol] = in[i]; } } } void 
MWCNNLayerImpl::padInput(float* THfVbcZJtANcLKxEriuV, int VCbcPxtPsBLTrHYdEvqn, int 
VenwEUlYwOBrwLVUhgUH, int UzaGmBLFEwmwaFXebUma, int jscBrjkVJyVfMMDjFpgl, int 
kMyEnepVyoNObTPqIpWo, int fvTCtkwXgyScJYogJVFU, int gNROjwaqhxDPvBWUCUcQ, float* 
jHaoHEqZgMiwRsdCogKz, int eFaDPmxDdzHlRYSAoMmX) { int shEncNmxJsMuJKwbrwok = 
(eFaDPmxDdzHlRYSAoMmX + 31) / 32 * 32; shEncNmxJsMuJKwbrwok = 
(shEncNmxJsMuJKwbrwok < 1024) ? shEncNmxJsMuJKwbrwok : 1024; int 
KHClOltUSuqFVVErSxVb = (eFaDPmxDdzHlRYSAoMmX + shEncNmxJsMuJKwbrwok - 
1) / shEncNmxJsMuJKwbrwok; padInputImpl<<<KHClOltUSuqFVVErSxVb, 
shEncNmxJsMuJKwbrwok>>>( THfVbcZJtANcLKxEriuV, VCbcPxtPsBLTrHYdEvqn, 
VenwEUlYwOBrwLVUhgUH, UzaGmBLFEwmwaFXebUma, jscBrjkVJyVfMMDjFpgl, kMyEnepVyoNObTPqIpWo, 
fvTCtkwXgyScJYogJVFU, gNROjwaqhxDPvBWUCUcQ, jHaoHEqZgMiwRsdCogKz, eFaDPmxDdzHlRYSAoMmX); } 
void __global__ __launch_bounds__(1024) fillOutputBufferImpl(signed char* in, 
int inputH, int inputW, int inputCh, int outputH, int outputW, int offsetH, int 
offsetW, signed char* out, int inputElems, int outputCh) { for (int i = 
blockDim.x * blockIdx.x + threadIdx.x; i < inputElems; i += blockDim.x * 
gridDim.x) { int idxB = i / (inputH * inputW * inputCh); int rem = (i - idxB * 
(inputH * inputW * inputCh)); int idxCh = rem / (inputH * inputW); int rem1 = 
rem - idxCh * (inputH * inputW); int idxH = rem1 / inputW; int idxCol = rem1 - 
idxH * inputW; if ((idxH < inputH) && (idxCol < inputW)) { int outputR = idxH + 
offsetH; int outputCol = idxCol + offsetW; *(out + idxB * (outputH * outputW * 
outputCh) + idxCh * (outputH * outputW) + outputR * (outputW) + outputCol) = 
*(in + i); } } } void MWCNNLayerImpl::fillOutputBuffer(signed char* 
THfVbcZJtANcLKxEriuV, int VCbcPxtPsBLTrHYdEvqn, int VenwEUlYwOBrwLVUhgUH, int 
UzaGmBLFEwmwaFXebUma, int jscBrjkVJyVfMMDjFpgl, int kMyEnepVyoNObTPqIpWo, int 
fvTCtkwXgyScJYogJVFU, int gNROjwaqhxDPvBWUCUcQ, signed char* jHaoHEqZgMiwRsdCogKz, int 
eFaDPmxDdzHlRYSAoMmX, int jhFUWlztBndwjbXwYNaJ) { int shEncNmxJsMuJKwbrwok 
= (eFaDPmxDdzHlRYSAoMmX < 1024) ? eFaDPmxDdzHlRYSAoMmX : 1024; int 
KHClOltUSuqFVVErSxVb = (eFaDPmxDdzHlRYSAoMmX + shEncNmxJsMuJKwbrwok - 
1) / shEncNmxJsMuJKwbrwok; 
fillOutputBufferImpl<<<KHClOltUSuqFVVErSxVb, shEncNmxJsMuJKwbrwok>>>( 
THfVbcZJtANcLKxEriuV, VCbcPxtPsBLTrHYdEvqn, VenwEUlYwOBrwLVUhgUH, UzaGmBLFEwmwaFXebUma, 
jscBrjkVJyVfMMDjFpgl, kMyEnepVyoNObTPqIpWo, fvTCtkwXgyScJYogJVFU, gNROjwaqhxDPvBWUCUcQ, 
jHaoHEqZgMiwRsdCogKz, eFaDPmxDdzHlRYSAoMmX, jhFUWlztBndwjbXwYNaJ); } void 
MWCNNLayerImpl::throwAllocationError(const int line, const char* file) { 
std::stringstream ss; ss << "Failed to allocate memory at line : " << line << 
", file : " << file << "\n"; throw std::runtime_error(ss.str()); } void 
MWCNNLayerImpl::setDescriptorForINT8(hipdnnTensorDescriptor_t& desc, 
MWTensor<signed char>* tensor, hipdnnDataType_t dataType, hipdnnTensorFormat_t 
dataFormat) { CUDNN_CALL(hipdnnSetTensor4dDescriptor(desc, dataFormat, dataType, 
tensor->getBatchSize(),  tensor->getChannels(),  tensor->getHeight(),  
tensor->getWidth()));  } } 