#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: predict1.cu
//
// GPU Coder version                    : 2.5
// CUDA/C/C++ source code generated on  : 31-Jul-2024 04:58:22
//

// Include Files
#include "predict1.h"
#include "DeepLearningNetwork.h"
#include "Predict_internal_types.h"
#include "MWCudaDimUtility.hpp"

// Function Declarations
static __global__ void
DeepLearningNetwork_predict_kernel1(const unsigned char varargin_1[60903],
                                    float inMiniBatchGroup_0_f1[60903]);

// Function Definitions
//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned char varargin_1[60903]
//                float inMiniBatchGroup_0_f1[60903]
// Return Type  : void
//
static __global__
    __launch_bounds__(512, 1) void DeepLearningNetwork_predict_kernel1(
        const unsigned char varargin_1[60903],
        float inMiniBatchGroup_0_f1[60903])
{
  unsigned long threadId;
  int i;
  int i1;
  int p;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  i = static_cast<int>(threadId % 101UL);
  threadId = (threadId - static_cast<unsigned long>(i)) / 101UL;
  i1 = static_cast<int>(threadId % 201UL);
  threadId = (threadId - static_cast<unsigned long>(i1)) / 201UL;
  p = static_cast<int>(threadId);
  if ((p < 3) && (i1 < 201) && (i < 101)) {
    inMiniBatchGroup_0_f1[(i + 101 * i1) + 20301 * p] =
        static_cast<float>(varargin_1[(i1 + 201 * i) + 20301 * p]);
  }
}

//
// Arguments    : decnet0_0 *obj
//                const unsigned char varargin_1[60903]
//                float varargout_1[13]
// Return Type  : void
//
namespace coder {
namespace internal {
void DeepLearningNetwork_predict(decnet0_0 *obj,
                                 const unsigned char varargin_1[60903],
                                 float varargout_1[13])
{
  float(*gpu_inMiniBatchGroup_0_f1)[60903];
  hipMalloc(&gpu_inMiniBatchGroup_0_f1, 243612UL);
  DeepLearningNetwork_predict_kernel1<<<dim3(119U, 1U, 1U),
                                        dim3(512U, 1U, 1U)>>>(
      varargin_1, *gpu_inMiniBatchGroup_0_f1);
  hipMemcpy(obj->getInputDataPointer(0), *gpu_inMiniBatchGroup_0_f1,
             obj->getLayerOutputSize(0, 0), hipMemcpyDeviceToDevice);
  obj->predict();
  hipMemcpy(varargout_1, obj->getLayerOutput(9, 0),
             obj->getLayerOutputSize(9, 0), hipMemcpyDeviceToDevice);
  hipFree(*gpu_inMiniBatchGroup_0_f1);
}

} // namespace internal
} // namespace coder

//
// File trailer for predict1.cu
//
// [EOF]
//
