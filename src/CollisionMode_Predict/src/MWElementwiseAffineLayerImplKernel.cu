#include "hip/hip_runtime.h"
#include "MWKernelHeaders.hpp"
#include <math.h>
#include <stdio.h>
 void __global__ __launch_bounds__(1024) scale_scalar_kernel(const float* 
inputBuffer, float* outputBuffer, float* olKGEIcsxmLSoMhRhEtP, long int 
YMNbgnUYZspjMLjwcIOS) { for (long int idx = blockDim.x * blockIdx.x + 
threadIdx.x; idx < YMNbgnUYZspjMLjwcIOS; idx += blockDim.x * gridDim.x) { 
outputBuffer[idx] = olKGEIcsxmLSoMhRhEtP[0] * inputBuffer[idx]; } } void __global__ 
__launch_bounds__(1024) scale_vector_kernel(const float* inputBuffer, float* 
outputBuffer, float* olKGEIcsxmLSoMhRhEtP, double YNmJhGSUszJKxsodxiuV, 
double YNDVziqpDddiXQKYZZhX, long int YMNbgnUYZspjMLjwcIOS) { for (long 
int idx = blockDim.x * blockIdx.x + threadIdx.x; idx < YMNbgnUYZspjMLjwcIOS; 
idx += blockDim.x * gridDim.x) { double batchIdx = floor(idx / 
YNDVziqpDddiXQKYZZhX); double i_batch = idx - (batchIdx * 
YNDVziqpDddiXQKYZZhX); double channelIdx = floor(i_batch / 
YNmJhGSUszJKxsodxiuV); outputBuffer[idx] = 
olKGEIcsxmLSoMhRhEtP[static_cast<long int>(channelIdx)] * inputBuffer[idx]; } } 
void __global__ __launch_bounds__(1024) scale_matrix2d_kernel(const float* 
inputBuffer, float* outputBuffer, float* olKGEIcsxmLSoMhRhEtP, double 
YNmJhGSUszJKxsodxiuV, long int YMNbgnUYZspjMLjwcIOS) { for (long int 
idx = blockDim.x * blockIdx.x + threadIdx.x; idx < YMNbgnUYZspjMLjwcIOS; idx += 
blockDim.x * gridDim.x) { double totalChannelIdx = floor(idx / 
YNmJhGSUszJKxsodxiuV); double i_channel = idx - (totalChannelIdx * 
YNmJhGSUszJKxsodxiuV); outputBuffer[idx] = 
olKGEIcsxmLSoMhRhEtP[static_cast<long int>(i_channel)] * inputBuffer[idx]; } } void 
__global__ __launch_bounds__(1024) scale_tensor3d_kernel(const float* 
inputBuffer, float* outputBuffer, float* olKGEIcsxmLSoMhRhEtP, double 
YNDVziqpDddiXQKYZZhX, long int YMNbgnUYZspjMLjwcIOS) { for (long int 
idx = blockDim.x * blockIdx.x + threadIdx.x; idx < YMNbgnUYZspjMLjwcIOS; idx += 
blockDim.x * gridDim.x) { double batchIdx = floor(idx / 
YNDVziqpDddiXQKYZZhX); double i_batch = idx - (batchIdx * 
YNDVziqpDddiXQKYZZhX); outputBuffer[idx] = 
olKGEIcsxmLSoMhRhEtP[static_cast<long int>(i_batch)] * inputBuffer[idx]; } } void 
__global__ __launch_bounds__(1024) offset_scalar_kernel(float* inputBuffer, 
float* outputBuffer, float* fhikqqlnUKCjleVKDqiG, long int YMNbgnUYZspjMLjwcIOS, 
bool ZKjSVYDDjACizBkGbqBq, int bERCRkGjpaKXMNComoYl, int 
unSXtdjDjpysqxmbIiPv) { for (long int idx = blockDim.x * blockIdx.x + 
threadIdx.x; idx < YMNbgnUYZspjMLjwcIOS; idx += blockDim.x * gridDim.x) { float 
out = inputBuffer[idx] + fhikqqlnUKCjleVKDqiG[0]; if (ZKjSVYDDjACizBkGbqBq) { out 
= out > unSXtdjDjpysqxmbIiPv ? unSXtdjDjpysqxmbIiPv : out; out = out < 
bERCRkGjpaKXMNComoYl ? bERCRkGjpaKXMNComoYl : out; } outputBuffer[idx] = out; 
} } void __global__ __launch_bounds__(1024) offset_vector_kernel(float* 
inputBuffer, float* outputBuffer, float* fhikqqlnUKCjleVKDqiG, double 
YNmJhGSUszJKxsodxiuV, double YNDVziqpDddiXQKYZZhX, long int 
YMNbgnUYZspjMLjwcIOS, bool ZKjSVYDDjACizBkGbqBq, int bERCRkGjpaKXMNComoYl, int 
unSXtdjDjpysqxmbIiPv) { for (long int idx = blockDim.x * blockIdx.x + 
threadIdx.x; idx < YMNbgnUYZspjMLjwcIOS; idx += blockDim.x * gridDim.x) { 
double batchIdx = floor(idx / YNDVziqpDddiXQKYZZhX); double i_batch = 
idx - (batchIdx * YNDVziqpDddiXQKYZZhX); double channelIdx = 
floor(i_batch / YNmJhGSUszJKxsodxiuV); float out = inputBuffer[idx] + 
fhikqqlnUKCjleVKDqiG[static_cast<long int>(channelIdx)]; if (ZKjSVYDDjACizBkGbqBq) 
{ out = out > unSXtdjDjpysqxmbIiPv ? unSXtdjDjpysqxmbIiPv : out; out = out < 
bERCRkGjpaKXMNComoYl ? bERCRkGjpaKXMNComoYl : out; } outputBuffer[idx] = out; 
} } void __global__ __launch_bounds__(1024) offset_matrix2d_kernel(float* 
inputBuffer, float* outputBuffer, float* fhikqqlnUKCjleVKDqiG, double 
YNmJhGSUszJKxsodxiuV, long int YMNbgnUYZspjMLjwcIOS, bool 
ZKjSVYDDjACizBkGbqBq, int bERCRkGjpaKXMNComoYl, int unSXtdjDjpysqxmbIiPv) { 
for (long int idx = blockDim.x * blockIdx.x + threadIdx.x; idx < 
YMNbgnUYZspjMLjwcIOS; idx += blockDim.x * gridDim.x) { double totalChannelIdx = 
floor(idx / YNmJhGSUszJKxsodxiuV); double i_channel = idx - 
(totalChannelIdx * YNmJhGSUszJKxsodxiuV); float out = 
inputBuffer[idx] + fhikqqlnUKCjleVKDqiG[static_cast<long int>(i_channel)]; if 
(ZKjSVYDDjACizBkGbqBq) { out = out > unSXtdjDjpysqxmbIiPv ? 
unSXtdjDjpysqxmbIiPv : out; out = out < bERCRkGjpaKXMNComoYl ? 
bERCRkGjpaKXMNComoYl : out; } outputBuffer[idx] = out; } } void __global__ 
__launch_bounds__(1024) offset_tensor3d_kernel(float* inputBuffer, float* 
outputBuffer, float* fhikqqlnUKCjleVKDqiG, double YNDVziqpDddiXQKYZZhX, 
long int YMNbgnUYZspjMLjwcIOS, bool ZKjSVYDDjACizBkGbqBq, int 
bERCRkGjpaKXMNComoYl, int unSXtdjDjpysqxmbIiPv) { for (long int idx = 
blockDim.x * blockIdx.x + threadIdx.x; idx < YMNbgnUYZspjMLjwcIOS; idx += 
blockDim.x * gridDim.x) { double batchIdx = floor(idx / 
YNDVziqpDddiXQKYZZhX); double i_batch = idx - (batchIdx * 
YNDVziqpDddiXQKYZZhX); float out = inputBuffer[idx] + 
fhikqqlnUKCjleVKDqiG[static_cast<long int>(i_batch)]; if (ZKjSVYDDjACizBkGbqBq) { 
out = out > unSXtdjDjpysqxmbIiPv ? unSXtdjDjpysqxmbIiPv : out; out = out < 
bERCRkGjpaKXMNComoYl ? bERCRkGjpaKXMNComoYl : out; } outputBuffer[idx] = out; 
} }