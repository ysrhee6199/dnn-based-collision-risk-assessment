#include "hip/hip_runtime.h"
#include "MWCudnnCommonHeaders.hpp"
#include "MWCudnnMaxPoolingLayerImpl.hpp"
#include <cmath>
 namespace MWCudnnTarget { void __global__ MWSetDyForBackPropImpl(float* 
OwscQfaoXJuSJFwXQahz, const int eqUIJyhXTwRqtPfXapcx); void __global__ 
doMWMaxPoolingLayerImpl(float* UVzBVEOIylFjkSgHwFMp, float* 
URgvgDXnZskIYGdtimcU, const int BRSPqxNffoBYKqpSVHne); 
MWMaxPoolingLayerImpl::MWMaxPoolingLayerImpl(MWCNNLayer* layer, 
MWTargetNetworkImpl* ntwk_impl, int DGzdAcREJHGXjyRzNjJV, int 
ECTnqgWHyHCHCLBZlffd, int FOcStuqCptsGIZXskVpC, int FpguQZSermqZCMRiUfML, 
int CTCbzQMDaLxINPbODdng, int CLOUhPjbgggWoXHTtmjC, int 
CpMjJjtGOeWOzwxpAAQP, int CqtPRJvHlGJFssiPzsOm, bool 
GDRXdUDklKFEYEfifhIH, int eqmVWbEcwBRGnVNDUtrG) : MWCNNLayerImpl(layer, 
ntwk_impl) , BLjrjqvCcCommiXWQLjs(GDRXdUDklKFEYEfifhIH) , 
UVzBVEOIylFjkSgHwFMp(0) , OwscQfaoXJuSJFwXQahz(0) , 
DCdZnqpcBnvXVgEsLBnz(DGzdAcREJHGXjyRzNjJV) , 
DqxLTLaJwwgQqmrtCDuu(ECTnqgWHyHCHCLBZlffd) , 
CufLFODQDXTAPyRqYodN(DGzdAcREJHGXjyRzNjJV) , 
DSsxcjIrUgZCKZovyNQf(ECTnqgWHyHCHCLBZlffd) , 
CGbFsczkgkhjcHoCKzBx(CTCbzQMDaLxINPbODdng) , 
CDJtexcMbXMWAmnNZsNf(CLOUhPjbgggWoXHTtmjC) , 
CZNYmBcNFSZWvaCklqeM(CpMjJjtGOeWOzwxpAAQP) , 
ClEhcJFlvGCgiavziIag(CqtPRJvHlGJFssiPzsOm) , 
FLuSVNoPhAFKtLUchSvv(FOcStuqCptsGIZXskVpC) , 
FeVcBgtQmTLtmnNcJGMY(FpguQZSermqZCMRiUfML) , 
puSFZkRJmyuFPfQRswDK(std::pow(2.0f, 
static_cast<float>(layer->getScalingExponent()))) , 
vFNECEAeLZsYsUxvlgqL(std::string{"INT8x4"}.compare(layer->getAccelMode()) 
== 0) , euppfEoiaoCTcVgRPVhA(eqmVWbEcwBRGnVNDUtrG) { 
CUDNN_CALL(hipdnnCreatePoolingDescriptor(&lWJYwWaFPmWNQDPrlqER)); 
createAndAddDescriptor(getLayer()->getOutputTensor(0)->getSourcePortIndex()); 
CUDNN_CALL(hipdnnCreateTensorDescriptor(&DRzwhbNPpftRRIXXfHzd)); } 
MWMaxPoolingLayerImpl::~MWMaxPoolingLayerImpl() { } void 
MWMaxPoolingLayerImpl::propagateSize() { MWTensorBase* ipTensorBase = 
getLayer()->getInputTensor(0); MWTensorBase* opTensorBase = 
getLayer()->getOutputTensor(0); if ((DCdZnqpcBnvXVgEsLBnz == -1) && 
(DqxLTLaJwwgQqmrtCDuu == -1)) { CufLFODQDXTAPyRqYodN = 
ipTensorBase->getHeight(); DSsxcjIrUgZCKZovyNQf = ipTensorBase->getWidth(); } int 
lXJKIOEATumoVKStGbVy = CGbFsczkgkhjcHoCKzBx; int 
leWFtIPrKkXLixGWBGJW = CZNYmBcNFSZWvaCklqeM; 
CUDNN_CALL(hipdnnSetPooling2dDescriptor( lWJYwWaFPmWNQDPrlqER, HIPDNN_POOLING_MAX, 
HIPDNN_NOT_PROPAGATE_NAN, CufLFODQDXTAPyRqYodN, DSsxcjIrUgZCKZovyNQf, 
lXJKIOEATumoVKStGbVy, leWFtIPrKkXLixGWBGJW, FLuSVNoPhAFKtLUchSvv, 
FeVcBgtQmTLtmnNcJGMY)); hipdnnTensorDescriptor_t* desc = 
getDescriptor(opTensorBase->getSourcePortIndex()); assert(desc); if 
(ipTensorBase->isInt8()) { if (vFNECEAeLZsYsUxvlgqL) { 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(DRzwhbNPpftRRIXXfHzd, 
HIPDNN_TENSOR_NCHW_VECT_C,  HIPDNN_DATA_INT8x4,  ipTensorBase->getBatchSize(),  
ipTensorBase->getChannels(),  ipTensorBase->getHeight(),  
ipTensorBase->getWidth())  ); MWCNNLayerImpl::setDescriptorForINT8(*desc, 
static_cast<MWTensor<signed char>*>(opTensorBase), HIPDNN_DATA_INT8x4, 
HIPDNN_TENSOR_NCHW_VECT_C); } else { CUDNN_CALL(hipdnnSetTensor4dDescriptor( 
DRzwhbNPpftRRIXXfHzd, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_INT8, 
ipTensorBase->getBatchSize(), ipTensorBase->getChannels(), 
ipTensorBase->getHeight(), ipTensorBase->getWidth())); 
MWCNNLayerImpl::setDescriptorForINT8(*desc, static_cast<MWTensor<signed 
char>*>(opTensorBase), HIPDNN_DATA_INT8, HIPDNN_TENSOR_NCHW); } } else { 
setDescriptor<float>(*desc, static_cast<MWTensor<float>*>(opTensorBase)); } } 
void MWMaxPoolingLayerImpl::allocate() { MWCNNLayer* maxpoolLayer = getLayer(); 
MWTensorBase* ipTensor = maxpoolLayer->getInputTensor(0); MWTensorBase* 
opTensor = maxpoolLayer->getOutputTensor(0); if (BLjrjqvCcCommiXWQLjs) { const 
int dMxIKDGTITyhdLqIHBLA = ipTensor->getNumElements(); CUDA_CALL( 
hipMalloc((void**)&UVzBVEOIylFjkSgHwFMp, sizeof(float) * 
dMxIKDGTITyhdLqIHBLA)); const int eqUIJyhXTwRqtPfXapcx = 
opTensor->getNumElements(); CUDA_CALL(hipMalloc((void**)&OwscQfaoXJuSJFwXQahz, 
sizeof(float) * eqUIJyhXTwRqtPfXapcx)); int shEncNmxJsMuJKwbrwok = 
(eqUIJyhXTwRqtPfXapcx < 1024) ? eqUIJyhXTwRqtPfXapcx : 1024; int 
KHClOltUSuqFVVErSxVb = (eqUIJyhXTwRqtPfXapcx + shEncNmxJsMuJKwbrwok - 
1) / shEncNmxJsMuJKwbrwok; 
MWSetDyForBackPropImpl<<<KHClOltUSuqFVVErSxVb, 
shEncNmxJsMuJKwbrwok>>>( OwscQfaoXJuSJFwXQahz, eqUIJyhXTwRqtPfXapcx); } } void 
MWMaxPoolingLayerImpl::deallocate() { if (UVzBVEOIylFjkSgHwFMp) { 
CUDA_FREE_CALL(UVzBVEOIylFjkSgHwFMp); UVzBVEOIylFjkSgHwFMp = 
NULL; } if (OwscQfaoXJuSJFwXQahz) { CUDA_FREE_CALL(OwscQfaoXJuSJFwXQahz); OwscQfaoXJuSJFwXQahz = 
NULL; } } void MWMaxPoolingLayerImpl::predict() { MWCNNLayer* maxpoolLayer = 
getLayer(); MWTensorBase* ipTensorBase = maxpoolLayer->getInputTensor(0); 
MWTensorBase* opTensorBase = maxpoolLayer->getOutputTensor(0); 
hipdnnTensorDescriptor_t* desc = 
getDescriptor(opTensorBase->getSourcePortIndex()); assert(desc); 
hipdnnTensorDescriptor_t XhAYHFyEVtlwoxGBuTpu; if (opTensorBase->isInt8()) { 
XhAYHFyEVtlwoxGBuTpu = DRzwhbNPpftRRIXXfHzd; MWTensor<signed char>* ipTensor = 
static_cast<MWTensor<signed char>*>(ipTensorBase); MWTensor<signed char>* 
opTensor = static_cast<MWTensor<signed char>*>(opTensorBase); 
CUDNN_CALL(hipdnnPoolingForward(*cRtIUoZRPICuQEOZOSzT->getCudnnHandle(), 
lWJYwWaFPmWNQDPrlqER, &puSFZkRJmyuFPfQRswDK, XhAYHFyEVtlwoxGBuTpu, 
ipTensor->getData(), getZeroPtr(), *desc, opTensor->getData())); } else { 
XhAYHFyEVtlwoxGBuTpu = MWCNNLayerImpl::getCuDNNDescriptor(ipTensorBase); 
MWTensor<float>* ipTensor = static_cast<MWTensor<float>*>(ipTensorBase); 
MWTensor<float>* opTensor = static_cast<MWTensor<float>*>(opTensorBase); 
CUDNN_CALL(hipdnnPoolingForward(*cRtIUoZRPICuQEOZOSzT->getCudnnHandle(), 
lWJYwWaFPmWNQDPrlqER, getOnePtr(), XhAYHFyEVtlwoxGBuTpu, ipTensor->getData(), 
getZeroPtr(), *desc, opTensor->getData())); if (BLjrjqvCcCommiXWQLjs) { 
CUDNN_CALL(hipdnnPoolingBackward( *cRtIUoZRPICuQEOZOSzT->getCudnnHandle(), 
lWJYwWaFPmWNQDPrlqER, getOnePtr(), *desc, opTensor->getData(), *desc, 
OwscQfaoXJuSJFwXQahz, XhAYHFyEVtlwoxGBuTpu, ipTensor->getData(), getZeroPtr(), 
XhAYHFyEVtlwoxGBuTpu, UVzBVEOIylFjkSgHwFMp)); int dMxIKDGTITyhdLqIHBLA = 
ipTensor->getNumElements(); int shEncNmxJsMuJKwbrwok = 
(dMxIKDGTITyhdLqIHBLA < 1024) ? dMxIKDGTITyhdLqIHBLA : 1024; int 
KHClOltUSuqFVVErSxVb = (dMxIKDGTITyhdLqIHBLA + shEncNmxJsMuJKwbrwok - 
1) / shEncNmxJsMuJKwbrwok; 
doMWMaxPoolingLayerImpl<<<KHClOltUSuqFVVErSxVb, 
shEncNmxJsMuJKwbrwok>>>( UVzBVEOIylFjkSgHwFMp, 
static_cast<MWTensor<float>*>(maxpoolLayer->getOutputTensor(1))->getData(), 
dMxIKDGTITyhdLqIHBLA); } } return; } void MWMaxPoolingLayerImpl::cleanup() { 
CUDNN_CALL(hipdnnDestroyPoolingDescriptor(lWJYwWaFPmWNQDPrlqER)); MWTensorBase* 
opTensorBase = getLayer()->getOutputTensor(0); if (opTensorBase->isInt8()) { 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(DRzwhbNPpftRRIXXfHzd)); } } float* 
MWMaxPoolingLayerImpl::getIndexData() { return 
static_cast<MWTensor<float>*>(getLayer()->getOutputTensor(1))->getData(); } 
void __global__ __launch_bounds__(1024) MWSetDyForBackPropImpl(float* 
OwscQfaoXJuSJFwXQahz, const int eqUIJyhXTwRqtPfXapcx) { for (int i = blockDim.x * 
blockIdx.x + threadIdx.x; i < eqUIJyhXTwRqtPfXapcx; i += blockDim.x * 
gridDim.x) { OwscQfaoXJuSJFwXQahz[i] = i + 1; } } void __global__ 
__launch_bounds__(1024) doMWMaxPoolingLayerImpl(float* 
UVzBVEOIylFjkSgHwFMp, float* URgvgDXnZskIYGdtimcU, const int 
BRSPqxNffoBYKqpSVHne) { for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < 
BRSPqxNffoBYKqpSVHne; i += blockDim.x * gridDim.x) { if 
(static_cast<int>(UVzBVEOIylFjkSgHwFMp[i]) != 0) { 
URgvgDXnZskIYGdtimcU[static_cast<int>(UVzBVEOIylFjkSgHwFMp[i]) - 1] = 
i; } } } } 