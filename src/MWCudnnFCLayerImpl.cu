#include "MWCudnnCommonHeaders.hpp"
#include "MWCudnnFCLayerImpl.hpp"
#include <vector>
 namespace MWCudnnTarget { MWFCLayerImpl::MWFCLayerImpl(MWCNNLayer* layer, 
MWTargetNetworkImpl* ntwk_impl, int XNZmftADYzuZnIYIpBaT, int 
kNsviQGMPdXzNMRixGWR, const char* xHViLEwTujGGrPZZgmbF, const char* 
JxwPQNPACGfmGpNncpCY) : MWCNNLayerImpl(layer, ntwk_impl) , 
BlRIQPyqJZORKENzSdYf(XNZmftADYzuZnIYIpBaT) , 
BuyZFXzwOMxcePIbCLfl(kNsviQGMPdXzNMRixGWR) , vIWQzNvYZSuxmOTVDFhU(NULL) , 
vpXxoeEhdEosLSsYXkNG(NULL) , IwKnaBoXVubIRYcxEJLH(NULL) , 
xHiBGayUfxIpXKkCTDNU(false) { 
CUDNN_CALL(hipdnnCreateTensorDescriptor(&JsZenQeBPMhwsyEhVHiD)); 
createAndAddDescriptor(getLayer()->getOutputTensor(0)->getSourcePortIndex()); 
CUDA_CALL(hipMalloc((void**)&vIWQzNvYZSuxmOTVDFhU, sizeof(float) * 
BlRIQPyqJZORKENzSdYf * BuyZFXzwOMxcePIbCLfl)); 
CUDA_CALL(hipMalloc((void**)&IwKnaBoXVubIRYcxEJLH, sizeof(float) * 
BuyZFXzwOMxcePIbCLfl)); vpXxoeEhdEosLSsYXkNG = 
MALLOC_CALL(sizeof(float) * BlRIQPyqJZORKENzSdYf * 
BuyZFXzwOMxcePIbCLfl); loadWeights(xHViLEwTujGGrPZZgmbF); 
loadBias(JxwPQNPACGfmGpNncpCY); } MWFCLayerImpl::~MWFCLayerImpl() { } void 
MWFCLayerImpl::propagateSize() { MWCNNLayer* fcLayer = getLayer(); 
MWTensorBase* opTensor = fcLayer->getOutputTensor(0); hipdnnTensorDescriptor_t* 
desc = getDescriptor(opTensor->getSourcePortIndex()); assert(desc); 
setDescriptor<float>(*desc, static_cast<MWTensor<float>*>(opTensor)); if 
(opTensor->getSequenceLength() == 1) { 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(JsZenQeBPMhwsyEhVHiD, HIPDNN_TENSOR_NCHW, 
HIPDNN_DATA_FLOAT, 1, BuyZFXzwOMxcePIbCLfl, 1, 1)); } else { int dims[5] 
= {1, 1, BuyZFXzwOMxcePIbCLfl, 1, 1}; int strides[5]; 
MWTensorBase::getStrides(dims, 5, strides); CUDNN_CALL( 
hipdnnSetTensorNdDescriptor(JsZenQeBPMhwsyEhVHiD, HIPDNN_DATA_FLOAT, 5, dims, 
strides)); } } void MWFCLayerImpl::loadWeights(const char* PtRNGuserCxHAQfyEjFc) 
{ FILE* QTXuPiGKeBUnmRzhlIDp = MWCNNLayer::openBinaryFile(PtRNGuserCxHAQfyEjFc); 
assert(QTXuPiGKeBUnmRzhlIDp); int cnEykmOGhLuyKuadExWe = BlRIQPyqJZORKENzSdYf * 
BuyZFXzwOMxcePIbCLfl;  MWCNNLayer::call_fread(vpXxoeEhdEosLSsYXkNG, 
sizeof(float), cnEykmOGhLuyKuadExWe, QTXuPiGKeBUnmRzhlIDp, PtRNGuserCxHAQfyEjFc); 
fclose(QTXuPiGKeBUnmRzhlIDp); } void MWFCLayerImpl::prepareWeights(float* 
wqggPBXZvtlxnxwngvAq) { int cnEykmOGhLuyKuadExWe = BlRIQPyqJZORKENzSdYf * 
BuyZFXzwOMxcePIbCLfl; MWCNNLayer* fcLayer = getLayer(); MWTensorBase* 
ipTensor = fcLayer->getInputTensor(0); if (ipTensor->getHeight() != 1 && 
ipTensor->getWidth() != 1) { float* KZWeXiYFmdpQdsgidKeG = 
MALLOC_CALL(sizeof(float) * ipTensor->getHeight() * ipTensor->getWidth()); for 
(int k = 0; k < cnEykmOGhLuyKuadExWe / ipTensor->getHeight() / ipTensor->getWidth(); 
k++) { for (int i = 0; i < ipTensor->getHeight() * ipTensor->getWidth(); i++) 
KZWeXiYFmdpQdsgidKeG[i] = wqggPBXZvtlxnxwngvAq[k * ipTensor->getHeight() * 
ipTensor->getWidth() + i]; for (int j = 0; j < ipTensor->getHeight(); j++) for 
(int i = 0; i < ipTensor->getWidth(); i++) wqggPBXZvtlxnxwngvAq[k * 
ipTensor->getHeight() * ipTensor->getWidth() + j * ipTensor->getWidth() + i] = 
KZWeXiYFmdpQdsgidKeG[j + i * ipTensor->getHeight()]; } free(KZWeXiYFmdpQdsgidKeG); } 
CUDA_CALL(hipMemcpy(vIWQzNvYZSuxmOTVDFhU, wqggPBXZvtlxnxwngvAq, sizeof(float) * 
cnEykmOGhLuyKuadExWe, hipMemcpyHostToDevice)); } void MWFCLayerImpl::loadBias(const 
char* PtRNGuserCxHAQfyEjFc) { MWCNNLayer* fcLayer = getLayer(); MWTensorBase* 
opTensor = fcLayer->getOutputTensor(0); FILE* QTXuPiGKeBUnmRzhlIDp = 
MWCNNLayer::openBinaryFile(PtRNGuserCxHAQfyEjFc); assert(QTXuPiGKeBUnmRzhlIDp); int 
cnEykmOGhLuyKuadExWe = BuyZFXzwOMxcePIbCLfl;  float* KZWeXiYFmdpQdsgidKeG = 
MALLOC_CALL(sizeof(float) * cnEykmOGhLuyKuadExWe); 
MWCNNLayer::call_fread(KZWeXiYFmdpQdsgidKeG, sizeof(float), cnEykmOGhLuyKuadExWe, 
QTXuPiGKeBUnmRzhlIDp, PtRNGuserCxHAQfyEjFc); CUDA_CALL(hipMemcpy(IwKnaBoXVubIRYcxEJLH, 
KZWeXiYFmdpQdsgidKeG, sizeof(float) * cnEykmOGhLuyKuadExWe, hipMemcpyHostToDevice)); 
free(KZWeXiYFmdpQdsgidKeG); fclose(QTXuPiGKeBUnmRzhlIDp); } void 
MWFCLayerImpl::setLearnables(std::vector<float*> learnables) { 
assert(learnables.size() == 2); float* wqggPBXZvtlxnxwngvAq = learnables[0]; 
prepareWeights(wqggPBXZvtlxnxwngvAq); float* JgLfgHrHMEMmMYTettJF = learnables[1]; 
CUDA_CALL(hipMemcpy(IwKnaBoXVubIRYcxEJLH, JgLfgHrHMEMmMYTettJF, sizeof(float) * 
BuyZFXzwOMxcePIbCLfl, hipMemcpyHostToDevice)); } void 
MWFCLayerImpl::postSetup() { if (!xHiBGayUfxIpXKkCTDNU) { 
prepareWeights(vpXxoeEhdEosLSsYXkNG); free(vpXxoeEhdEosLSsYXkNG); 
vpXxoeEhdEosLSsYXkNG = NULL; xHiBGayUfxIpXKkCTDNU = true; } } void 
MWFCLayerImpl::predict() { MWCNNLayer* fcLayer = getLayer(); MWTensorBase* 
ipTensorBase = fcLayer->getInputTensor(0); MWTensorBase* opTensorBase = 
fcLayer->getOutputTensor(0); MWTensor<float>* ipTensor = 
static_cast<MWTensor<float>*>(ipTensorBase); MWTensor<float>* opTensor = 
static_cast<MWTensor<float>*>(opTensorBase); int numOutputRows = 
opTensor->getChannels(); int numOutputCols = ipTensor->getBatchSize() * 
ipTensor->getSequenceLength(); int innerDimension = ipTensor->getHeight() * 
ipTensor->getWidth() * ipTensor->getChannels(); int UEESbUvbMihFnquvuFij = 1; int 
UKtMXCCqdjeyaVHabkxg = 1; if (opTensor->getBatchSize() == 1 && 
opTensor->getSequenceLength() == 1) { CUDA_CALL(hipMemcpy(opTensor->getData(), 
IwKnaBoXVubIRYcxEJLH, sizeof(float) * numOutputRows, hipMemcpyDeviceToDevice)); 
CUBLAS_CALL(hipblasSgemv(*cRtIUoZRPICuQEOZOSzT->getCublasHandle(), HIPBLAS_OP_T, 
innerDimension, numOutputRows, getOnePtr(), vIWQzNvYZSuxmOTVDFhU, innerDimension, 
ipTensor->getData(), UEESbUvbMihFnquvuFij, getOnePtr(), opTensor->getData(), 
UKtMXCCqdjeyaVHabkxg)); } else { 
CUBLAS_CALL(hipblasSgemm(*cRtIUoZRPICuQEOZOSzT->getCublasHandle(), HIPBLAS_OP_T, 
HIPBLAS_OP_N, numOutputRows, numOutputCols, innerDimension, getOnePtr(), 
vIWQzNvYZSuxmOTVDFhU, innerDimension, ipTensor->getData(), innerDimension, 
getZeroPtr(), opTensor->getData(), numOutputRows)); hipdnnTensorDescriptor_t* 
desc = getDescriptor(opTensor->getSourcePortIndex()); assert(desc); 
CUDNN_CALL(hipdnnAddTensor(*cRtIUoZRPICuQEOZOSzT->getCudnnHandle(), getOnePtr(), 
JsZenQeBPMhwsyEhVHiD, IwKnaBoXVubIRYcxEJLH, getOnePtr(), *desc, opTensor->getData())); } 
return; } void MWFCLayerImpl::cleanup() { if (vIWQzNvYZSuxmOTVDFhU) { 
CUDA_FREE_CALL(vIWQzNvYZSuxmOTVDFhU); vIWQzNvYZSuxmOTVDFhU = NULL; } 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(JsZenQeBPMhwsyEhVHiD)); if 
(IwKnaBoXVubIRYcxEJLH) { CUDA_FREE_CALL(IwKnaBoXVubIRYcxEJLH); IwKnaBoXVubIRYcxEJLH = NULL; } } } 