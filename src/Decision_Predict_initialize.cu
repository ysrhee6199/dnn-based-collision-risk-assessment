//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: Decision_Predict_initialize.cu
//
// GPU Coder version                    : 2.5
// CUDA/C/C++ source code generated on  : 11-Jun-2024 01:16:03
//

// Include Files
#include "Decision_Predict_initialize.h"
#include "Decision_Predict.h"
#include "Decision_Predict_data.h"

// Function Definitions
//
// Arguments    : void
// Return Type  : void
//
void Decision_Predict_initialize()
{
  Decision_Predict_init();
  hipGetLastError();
  isInitialized_Decision_Predict = true;
}

//
// File trailer for Decision_Predict_initialize.cu
//
// [EOF]
//
