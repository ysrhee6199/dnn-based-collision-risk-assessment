#include "MWKernelHeaders.hpp"
#include "MWCudnnCommonHeaders.hpp"
#include "MWCudnnElementwiseAffineLayerImpl.hpp"
#include <cmath>
#include <cassert>
#include <cstdio>
 namespace MWCudnnTarget { 
MWElementwiseAffineLayerImpl::MWElementwiseAffineLayerImpl(MWCNNLayer* layer, 
MWTargetNetworkImpl* ntwk_impl, int scale_H, int scale_W, int scale_C, int 
offset_H, int offset_W, int offset_C, bool isClipped, int lowerbound, int 
upperbound, const char* pdleXafalaHAmketaFyq, const char* 
gzSTokDHvkXefhiGDcWL) : MWCNNLayerImpl(layer, ntwk_impl) , 
olKGEIcsxmLSoMhRhEtP(NULL) , fhikqqlnUKCjleVKDqiG(NULL) , pFoPPXxxFRbjXXxQWItv(scale_H) 
, pckLLTEdVPoCZLRwyDnM(scale_W) , osBZbKVTgXwTSsGSbdth(scale_C) , 
gWETwFdWHfKuelmlKNCC(offset_H) , gsJtSpgIkTNvahoTFqow(offset_W) , 
gTcJMwtYuwiqqUmqvKhT(offset_C) , ZKjSVYDDjACizBkGbqBq(isClipped) , 
bERCRkGjpaKXMNComoYl(lowerbound) , unSXtdjDjpysqxmbIiPv(upperbound) { 
CUDA_CALL(hipMalloc((void**)&olKGEIcsxmLSoMhRhEtP, sizeof(float) * 
pFoPPXxxFRbjXXxQWItv * pckLLTEdVPoCZLRwyDnM * osBZbKVTgXwTSsGSbdth)); 
CUDA_CALL(hipMalloc( (void**)&fhikqqlnUKCjleVKDqiG, sizeof(float) * 
gWETwFdWHfKuelmlKNCC * gsJtSpgIkTNvahoTFqow * gTcJMwtYuwiqqUmqvKhT)); 
loadScale(pdleXafalaHAmketaFyq); loadOffset(gzSTokDHvkXefhiGDcWL); } 
MWElementwiseAffineLayerImpl::~MWElementwiseAffineLayerImpl() { } void 
MWElementwiseAffineLayerImpl::propagateSize() { } void 
MWElementwiseAffineLayerImpl::predict() { MWTensorBase* ipTensorBase = 
getLayer()->getInputTensor(0); MWTensorBase* opTensorBase = 
getLayer()->getOutputTensor(0); MWTensor<float>* ipTensor = 
static_cast<MWTensor<float>*>(ipTensorBase); MWTensor<float>* opTensor = 
static_cast<MWTensor<float>*>(opTensorBase); int WmXADZOqdcQvtBUvFerh = 
ipTensor->getHeight(); int WprSrhAStKGxyXeoxETy = ipTensor->getWidth(); int 
WerBmCOBWhvoFbdqfitc = ipTensor->getChannels(); long int 
YNmJhGSUszJKxsodxiuV = WmXADZOqdcQvtBUvFerh * WprSrhAStKGxyXeoxETy; long 
int YNDVziqpDddiXQKYZZhX = YNmJhGSUszJKxsodxiuV * 
WerBmCOBWhvoFbdqfitc; long int YMNbgnUYZspjMLjwcIOS = ipTensor->getNumElements(); 
long int rlQsibXJSWJVnUVpdNeL = ((YMNbgnUYZspjMLjwcIOS + 31) / 32) * 32; int 
shEncNmxJsMuJKwbrwok = (rlQsibXJSWJVnUVpdNeL < 1024) ? rlQsibXJSWJVnUVpdNeL : 1024; 
long int KHClOltUSuqFVVErSxVb = (YMNbgnUYZspjMLjwcIOS + 
shEncNmxJsMuJKwbrwok - 1) / shEncNmxJsMuJKwbrwok; long int 
pbePKOGQbvmzToFbiRkR = pFoPPXxxFRbjXXxQWItv * pckLLTEdVPoCZLRwyDnM * 
osBZbKVTgXwTSsGSbdth; long int gcGbhKACQPAogUYXHedj = gWETwFdWHfKuelmlKNCC * 
gsJtSpgIkTNvahoTFqow * gTcJMwtYuwiqqUmqvKhT; assert(pbePKOGQbvmzToFbiRkR <= 
YMNbgnUYZspjMLjwcIOS); assert(gcGbhKACQPAogUYXHedj <= YMNbgnUYZspjMLjwcIOS); if 
(pbePKOGQbvmzToFbiRkR == 1) { scale_scalar_kernel<<<KHClOltUSuqFVVErSxVb, 
shEncNmxJsMuJKwbrwok>>>( ipTensor->getData(), opTensor->getData(), 
olKGEIcsxmLSoMhRhEtP, YMNbgnUYZspjMLjwcIOS); } else if (pFoPPXxxFRbjXXxQWItv == 1 && 
pckLLTEdVPoCZLRwyDnM == 1 && pbePKOGQbvmzToFbiRkR > 1) { 
scale_vector_kernel<<<KHClOltUSuqFVVErSxVb, shEncNmxJsMuJKwbrwok>>>( 
ipTensor->getData(), opTensor->getData(), olKGEIcsxmLSoMhRhEtP, 
YNmJhGSUszJKxsodxiuV, YNDVziqpDddiXQKYZZhX, 
YMNbgnUYZspjMLjwcIOS); } else if (YNDVziqpDddiXQKYZZhX == 
pbePKOGQbvmzToFbiRkR) { scale_tensor3d_kernel<<<KHClOltUSuqFVVErSxVb, 
shEncNmxJsMuJKwbrwok>>>( ipTensor->getData(), opTensor->getData(), 
olKGEIcsxmLSoMhRhEtP, YNDVziqpDddiXQKYZZhX, YMNbgnUYZspjMLjwcIOS); } else { 
scale_matrix2d_kernel<<<KHClOltUSuqFVVErSxVb, 
shEncNmxJsMuJKwbrwok>>>( ipTensor->getData(), opTensor->getData(), 
olKGEIcsxmLSoMhRhEtP, YNmJhGSUszJKxsodxiuV, YMNbgnUYZspjMLjwcIOS); } if 
(gcGbhKACQPAogUYXHedj == 1) { offset_scalar_kernel<<<KHClOltUSuqFVVErSxVb, 
shEncNmxJsMuJKwbrwok>>>( opTensor->getData(), opTensor->getData(), 
fhikqqlnUKCjleVKDqiG, YMNbgnUYZspjMLjwcIOS, ZKjSVYDDjACizBkGbqBq, 
bERCRkGjpaKXMNComoYl, unSXtdjDjpysqxmbIiPv); } else if (gWETwFdWHfKuelmlKNCC 
== 1 && gsJtSpgIkTNvahoTFqow == 1 && gcGbhKACQPAogUYXHedj > 1) { 
offset_vector_kernel<<<KHClOltUSuqFVVErSxVb, shEncNmxJsMuJKwbrwok>>>( 
opTensor->getData(), opTensor->getData(), fhikqqlnUKCjleVKDqiG, 
YNmJhGSUszJKxsodxiuV, YNDVziqpDddiXQKYZZhX, 
YMNbgnUYZspjMLjwcIOS, ZKjSVYDDjACizBkGbqBq, bERCRkGjpaKXMNComoYl, 
unSXtdjDjpysqxmbIiPv); } else if (YNDVziqpDddiXQKYZZhX == 
gcGbhKACQPAogUYXHedj) { offset_tensor3d_kernel<<<KHClOltUSuqFVVErSxVb, 
shEncNmxJsMuJKwbrwok>>>( opTensor->getData(), opTensor->getData(), 
fhikqqlnUKCjleVKDqiG, YNDVziqpDddiXQKYZZhX, YMNbgnUYZspjMLjwcIOS, 
ZKjSVYDDjACizBkGbqBq, bERCRkGjpaKXMNComoYl, unSXtdjDjpysqxmbIiPv); } else { 
offset_matrix2d_kernel<<<KHClOltUSuqFVVErSxVb, 
shEncNmxJsMuJKwbrwok>>>( opTensor->getData(), opTensor->getData(), 
fhikqqlnUKCjleVKDqiG, YNmJhGSUszJKxsodxiuV, YMNbgnUYZspjMLjwcIOS, 
ZKjSVYDDjACizBkGbqBq, bERCRkGjpaKXMNComoYl, unSXtdjDjpysqxmbIiPv); } return; 
} void MWElementwiseAffineLayerImpl::cleanup() { if (olKGEIcsxmLSoMhRhEtP) { 
CUDA_FREE_CALL(olKGEIcsxmLSoMhRhEtP); olKGEIcsxmLSoMhRhEtP = NULL; } if 
(fhikqqlnUKCjleVKDqiG) { CUDA_FREE_CALL(fhikqqlnUKCjleVKDqiG); fhikqqlnUKCjleVKDqiG = 
NULL; } } void MWElementwiseAffineLayerImpl::loadScale(const char* 
pdleXafalaHAmketaFyq) { FILE* QTXuPiGKeBUnmRzhlIDp = 
MWCNNLayer::openBinaryFile(pdleXafalaHAmketaFyq); assert(QTXuPiGKeBUnmRzhlIDp); long 
int cnEykmOGhLuyKuadExWe = pFoPPXxxFRbjXXxQWItv * pckLLTEdVPoCZLRwyDnM * 
osBZbKVTgXwTSsGSbdth; float* KZWeXiYFmdpQdsgidKeG = MALLOC_CALL(sizeof(float) * 
cnEykmOGhLuyKuadExWe); MWCNNLayer::call_fread(KZWeXiYFmdpQdsgidKeG, sizeof(float), 
cnEykmOGhLuyKuadExWe, QTXuPiGKeBUnmRzhlIDp, pdleXafalaHAmketaFyq); 
CUDA_CALL(hipMemcpy(olKGEIcsxmLSoMhRhEtP, KZWeXiYFmdpQdsgidKeG, sizeof(float) * 
cnEykmOGhLuyKuadExWe, hipMemcpyHostToDevice)); free(KZWeXiYFmdpQdsgidKeG); 
fclose(QTXuPiGKeBUnmRzhlIDp); } void MWElementwiseAffineLayerImpl::loadOffset(const 
char* gzSTokDHvkXefhiGDcWL) { FILE* QTXuPiGKeBUnmRzhlIDp = 
MWCNNLayer::openBinaryFile(gzSTokDHvkXefhiGDcWL); assert(QTXuPiGKeBUnmRzhlIDp); long 
int cnEykmOGhLuyKuadExWe = gWETwFdWHfKuelmlKNCC * gsJtSpgIkTNvahoTFqow * 
gTcJMwtYuwiqqUmqvKhT; float* KZWeXiYFmdpQdsgidKeG = MALLOC_CALL(sizeof(float) * 
cnEykmOGhLuyKuadExWe); MWCNNLayer::call_fread(KZWeXiYFmdpQdsgidKeG, sizeof(float), 
cnEykmOGhLuyKuadExWe, QTXuPiGKeBUnmRzhlIDp, gzSTokDHvkXefhiGDcWL); 
CUDA_CALL(hipMemcpy(fhikqqlnUKCjleVKDqiG, KZWeXiYFmdpQdsgidKeG, sizeof(float) * 
cnEykmOGhLuyKuadExWe, hipMemcpyHostToDevice)); free(KZWeXiYFmdpQdsgidKeG); 
fclose(QTXuPiGKeBUnmRzhlIDp); } } 